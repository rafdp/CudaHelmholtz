#include "hip/hip_runtime.h"


#include "CudaCalc.h"

#define _PI 3.1415926f

__global__ void BornForRecieversKernel (int * P_recv, InputData_t* INPUT_DATA_PTR);

__global__ void DevicePrintData (InputDataOnDevice * inputDataPtr);

__global__ void DevicePrint ();

__device__ thrust::complex<float> * UiPtr;

__device__ Point3DDevice_t <float> * PointsPtr;

__device__ InputDataOnDevice * inputDataPtr;


	template <typename T>
	__host__ __device__	
	Point3DDevice_t<T>::Point3DDevice_t (const Point3D_t &copy)
    {
        x = (T) copy.x;
        y = (T) copy.y;
        z = (T) copy.z;
    }
   
	template <typename T>
    __host__ __device__
    Point3DDevice_t<T>::Point3DDevice_t (): x(0), y(0), z (0){}

	template <typename T>    
	template <typename T1>
    __host__ __device__
    Point3DDevice_t<T>::Point3DDevice_t (T1 tx, T1 ty, T1 tz) : 
        x (tx), y (ty), z (tz){}

	template <typename T>
    __host__ __device__
    Point3DDevice_t<T>::Point3DDevice_t (T* begin) : 
        x (begin [0]), y (begin [1]), z (begin [2]){}

    template <typename T>
    __host__ __device__
    T Point3DDevice_t<T>::len () const
    {
        return (T) sqrtf (x*x + y*y + z*z);
    }

struct BornCalculation  // rewrite inputdata use
{
	const Point3DDevice_t <float> rj;
	BornCalculation(Point3D_t _rj) : rj(_rj) {}

    __device__
	thrust::complex<float> operator()(thrust::complex<float> idxz) const
	{

		int idx = roundf(idxz.real());		
		InputDataOnDevice* d_inputData = inputDataPtr;
	
		Point3DDevice_t <float> r = *(PointsPtr + idx);

		Point3DDevice_t <float> dr = {r.x - rj.x, r.y - rj.y, r.z - rj.z};
						            
		return *(UiPtr + idx)  * thrust::exp(d_inputData -> uiCoeff_ * dr.len()) / (4 * _PI * dr.len());
	}
};

struct PrintComplexVector
{
  __device__
  void operator () (thrust::complex<float> val)
  {     
	printf ("%e + %ei\n", val.real(), val.imag());
  }
};

struct PrintPointsVector
{
  __device__
  void operator () (const Point3DDevice_t <float> &val)
  {    
	printf ("(%f, %f, %f)\n", val.x, val.y, val.z);
  }
};


struct complexPlus
{
    __host__ __device__
    thrust::complex <float> operator () (const thrust::complex<float> &z1, const thrust::complex<float> &z2) const
    {
        return z1 + z2;
    }
};

struct ComplexIndex
{
	__host__ __device__
	thrust::complex <float> operator () (int index) const
	{
		return thrust::complex <float> (index * 1.0f, 0.0f);
	}
};

struct UiMultiply
{
	__device__
	thrust::complex <float> operator()(const thrust::complex<float>& ds, const Point3DDevice_t<float>& r) const 
	{
		//printf ("%f\n",	ds.imag());	
		if (abs (r.len()) > 0.0000001) return inputDataPtr -> w2h3_ * ds * thrust::exp(inputDataPtr -> uiCoeff_ * r.len()) / (4 * _PI * r.len());
	}

};

struct IndexFromSequence
{
    __device__
    Point3DDevice_t<float> operator() (int idx) const
    {

        Point3DDevice_t<float> point = { 1.0f * (idx % inputDataPtr->size1_),
                                         1.0f * ((idx / inputDataPtr->size1_) % inputDataPtr->discretizationSize_.y),
                                         1.0f * (idx / inputDataPtr->size2_)};
        point = {(float) (point.x*inputDataPtr->discreteBlockSize_.x*1.0f +
                 inputDataPtr->anomalyPos_.x +
                 inputDataPtr->discreteBlockSize_.x / 2.0),
                 (float) (point.y*inputDataPtr->discreteBlockSize_.y*1.0f +
                 inputDataPtr->anomalyPos_.y +
                 inputDataPtr->discreteBlockSize_.y / 2.0),
                 (float) (point.z*inputDataPtr->discreteBlockSize_.z*1.0f +
                 inputDataPtr->anomalyPos_.z +
                 inputDataPtr->discreteBlockSize_.z / 2.0)};
        return point;
    }
};

__global__

void DevicePrint ()
{
    printf ("--------------------------------------------------------------\n");
    printf ("threadIdx.x: %d\n", threadIdx.x);
    printf ("--------------------------------------------------------------\n");
}


__global__ void DevicePrintData ()
{
    /*    Point3DDevice_t<float> sourcePos_;
    float w_; //DROP
    thrust::complex<float> uiCoeff_;
    Point3DDevice_t<float> anomalyPos_;
    Point3DDevice_t<float> anomalySize_;
    Point3DDevice_t<int>   discretizationSize_;
    Point3DDevice_t<int>   discreteBlockSize_;
    int                    size3_;
    int                    size2_;
    int                    size1_;
    float                  w2h3_;*/

    printf ("device address %x\n", inputDataPtr);

    InputDataOnDevice& handle = *(inputDataPtr);

    printf ("--------------------------------------------------------------\n");
    printf ("Printing from device:\n");
    printf ("   sourcePos_: %f %f %f\n",
            inputDataPtr->sourcePos_.x,
            handle.sourcePos_.y,
            handle.sourcePos_.z);
    printf ("   w: %f\n",
            handle.w_);
    printf ("   uiCoeff: %f i*%f\n",
            handle.uiCoeff_.real (),
            handle.uiCoeff_.imag ());
    printf ("   anomalySize_: %f %f %f\n",
            handle.anomalySize_.x,
            handle.anomalySize_.y,
            handle.anomalySize_.z);
    printf ("   discretizationSize_: %d %d %d\n",
            handle.discretizationSize_.x,
            handle.discretizationSize_.y,
            handle.discretizationSize_.z);
    printf ("   discreteBlockSize_: %d %d %d\n",
            handle.discreteBlockSize_.x,
            handle.discreteBlockSize_.y,
            handle.discreteBlockSize_.z);
    printf ("   size3_: %d %d %d\n", handle.size3_);
    printf ("   size2_: %d %d %d\n", handle.size2_);
    printf ("   size1_: %d %d %d\n", handle.size1_);
    printf ("   w2h3_: %f\n", handle.w2h3_);
    printf ("End print from device\n");
    printf ("--------------------------------------------------------------\n");
}

extern "C"
void ExternalKernelCaller (InputData_t* inputDataPtr_, std::vector<std::complex<float> >* retData)
{

	

	InputData_t& inputData = *inputDataPtr_;
	InputDataOnDevice* deviceInputData = nullptr;

//	printf ("ERROR: %s\n", hipGetErrorString(hipMalloc ((void**) &deviceInputData, sizeof (InputDataOnDevice))));
//                                                                       +-------inputDataPtr
//                                                                       |
//                                                                       v
//    	printf ("ERROR: %s\n", hipGetErrorString(hipMemcpyToSymbol(HIP_SYMBOL(&inputDataPtr),
  //                                                               &deviceInputData,
    //                                                             sizeof(InputDataOnDevice*))));
    printf("ERROR: %s\n", hipMalloc ((void**) &deviceInputData, sizeof (InputDataOnDevice)));

    printf("ERROR: %s\n", hipMemcpyToSymbol(HIP_SYMBOL(inputDataPtr), &deviceInputData, sizeof(InputDataOnDevice*)));

	int recvNum = inputData.Nreceivers_;
	


    	int size3 = inputData.discretizationSize_[0] *
                inputData.discretizationSize_[1] *
                inputData.discretizationSize_[2];



    	InputDataOnDevice hostDataCopy = {(inputData.sourcePos_),
                                      (float) (2*3.141592f*inputData.f_),
                                      thrust::complex<float> (0, (float) (2*3.141592f*inputData.f_/inputData.c_)),
                                     (inputData.anomalyPos_),
                                      (inputData.anomalySize_),
                                      inputData.discretizationSize_,
                                      inputData.discreteBlockSize_,
                                      size3,
                                      inputData.discretizationSize_[0] *
                                      inputData.discretizationSize_[1],
                                      inputData.discretizationSize_[0],
                                      (float)(4*3.1415926f*3.1415926f*inputData.f_*inputData.f_*
                                      inputData.discreteBlockSize_[0]*inputData.discreteBlockSize_[1]*inputData.discreteBlockSize_[2])};


    hipMemcpy (deviceInputData, &hostDataCopy, sizeof (InputDataOnDevice), hipMemcpyHostToDevice);

    printf ("About to call kernel\n");
    DevicePrintData<<<1, 1>>> ();
    hipDeviceSynchronize ();
    printf ("Kernel returned\n");
	


    thrust::host_vector<thrust::complex<float> > hostDs2Matrix (size3);

    for (int i = 0; i < size3; i++)
    {
    	hostDs2Matrix[i] = thrust::complex <float> (float (inputData.ds2_[i]), 0.0);
	//printf ("%e + 0\n", hostDs2Matrix[i].real());

    }

	printf ("hostDs2 done\n");
    	thrust::device_vector<thrust::complex<float> > dS (hostDs2Matrix);
	//thrust::for_each (dS.begin(), dS.begin() + 20, PrintComplexVector());
	printf ("ds2 sent to device\n");

	thrust::device_vector <thrust::complex <float> > Ui (size3);
	 void * tempPtr = Ui.data ().get ();
   hipMemcpyToSymbol(HIP_SYMBOL(UiPtr),
                       &tempPtr,
                      sizeof(void*));


	thrust::device_vector <Point3DDevice_t <float> > Points (size3);
	tempPtr = Points.data ().get ();
    hipMemcpyToSymbol(HIP_SYMBOL(PointsPtr),
                       &tempPtr,
                       sizeof(void*));
	printf ("arrays copied\n");
	
	
	
	thrust::tabulate(Points.begin(), Points.end(), IndexFromSequence()); // filling Point with coordinates
//hipDeviceSynchronize ();
	printf ("tabulated\n");
	//PrintPointsVector printP;
	//thrust::for_each (Points.begin(), Points.end(), printP);
	hipDeviceSynchronize ();
	printf ("afte forech\n");

	////////////////////////
	hipEvent_t start, stop;
	float time;
	hipEventCreate(&start);
	hipEventRecord(start, 0);
	hipEventCreate(&stop);
        ////////////////////////

	thrust::transform(dS.begin(), dS.end(), Points.begin(), Ui.begin(), UiMultiply()); // filling Ui array with w^2 * G(r) * ds^2 * h^3
    	hipDeviceSynchronize ();
	printf ("transformed\n");
	//PrintComplexVector printC;
	//thrust::for_each (Ui.begin(), Ui.begin() + 20, printC);
    	hipDeviceSynchronize ();

	thrust::device_vector <thrust::complex <float> > d_output(recvNum);
	

	for (int i = 0; i < recvNum; i ++)
	{
		Point3D_t rj = inputData.receivers_[i];
		//printf ("started counting recv n %d\n", i);

		thrust::device_vector <thrust::complex<float> > BornForReciever(size3);
		thrust::complex <float> init = (0.0f, 0.0f);
		thrust::tabulate(BornForReciever.begin(), BornForReciever.end(), ComplexIndex()); 

		//float init = 0; //ui to global
		complexPlus binary_op;

		(*retData) [i] = thrust::transform_reduce(BornForReciever.begin(), BornForReciever.end(), BornCalculation (rj), init, binary_op); //born calc to global ui
	}
	//////////////////////////////////////////////
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);
	printf ("Time for the kernel: %f ms\n", time);
	//////////////////////////////////////////////

}

