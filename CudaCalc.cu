#include "hip/hip_runtime.h"

//=================================================================

#include <thrust/device_vector.h>
#include <thrust/transform.h>
#include <thrust/sequence.h>
#include <thrust/copy.h>
#include <thrust/fill.h>
#include <thrust/replace.h>
#include <thrust/transform_reduce.h>
#include <thrust/memory.h>
#include <thrust/complex.h>
#include <thrust/device_new.h>
#include <thrust/functional.h>
#include "hipblas.h"


#include "CudaCalc.h"

struct InputDataOnDevice
{
    Point3DDevice_t<float> sourcePos_;
    float w_; //DROP
    thrust::complex<float> uiCoeff_;
    Point3DDevice_t<float> anomalyPos_;
    Point3DDevice_t<float> anomalySize_;
    Point3DDevice_t<int>   discretizationSize_;
    Point3DDevice_t<int>   discreteBlockSize_;
    int                    size3_;
    int                    size2_;
    int                    size1_;
    float                  w2h3_;
};

//-----------------------------------------------------------------

__global__ void BornForRecieversKernel (complex<double> * P_recv, InputData_t* INPUT_DATA_PTR)
//cannot pass host data pointer to kernel, need to copy to device memory
{
    /*Point3D_t r = {static_cast<int> (threadIdx.x + blockIdx.x * BLOCK_SIZE_),
                   static_cast<int> (threadIdx.y + blockIdx.y * BLOCK_SIZE_),
                   static_cast<int> (threadIdx.z + blockIdx.z * BLOCK_SIZE_)};
//static cast needed for unsigned int -> int warning

    int recv_num = INPUT_DATA_PTR->recievers_.size ();
    for (int i = 0; i < recv_num; i ++)
    {
        P_recv [i] += BornForPoint (r, INPUT_DATA_PTR->recievers_ [i]);
    }*/
//There is a major problem with this kernel code: CUDA cannot call functions that are implemented on host (e.g. BornForPoint).
//You need to rewrite them on the gpu via cuda

}

__global__ void DevicePrint ()
{
    printf ("--------------------------------------------------------------\n");
    printf ("threadIdx.x: %d\n", threadIdx.x);
    printf ("--------------------------------------------------------------\n");
}


__global__ void DevicePrintData (InputDataOnDevice * inputDataPtr)
{
    /*    Point3DDevice_t<float> sourcePos_;
    float w_; //DROP
    thrust::complex<float> uiCoeff_;
    Point3DDevice_t<float> anomalyPos_;
    Point3DDevice_t<float> anomalySize_;
    Point3DDevice_t<int>   discretizationSize_;
    Point3DDevice_t<int>   discreteBlockSize_;
    int                    size3_;
    int                    size2_;
    int                    size1_;
    float                  w2h3_;*/

    printf ("device address %x\n", inputDataPtr);

    InputDataOnDevice& handle = *(inputDataPtr);

    printf ("--------------------------------------------------------------\n");
    printf ("Printing from device:\n");
    printf ("   sourcePos_: %f %f %f\n",
            inputDataPtr->sourcePos_.x,
            handle.sourcePos_.y,
            handle.sourcePos_.z);
    printf ("   w: %f\n",
            handle.w_);
    printf ("   uiCoeff: %f i*%f\n",
            handle.uiCoeff_.real (),
            handle.uiCoeff_.imag ());
    printf ("   anomalySize_: %f %f %f\n",
            handle.anomalySize_.x,
            handle.anomalySize_.y,
            handle.anomalySize_.z);
    printf ("   discretizationSize_: %d %d %d\n",
            handle.discretizationSize_.x,
            handle.discretizationSize_.y,
            handle.discretizationSize_.z);
    printf ("   discreteBlockSize_: %d %d %d\n",
            handle.discreteBlockSize_.x,
            handle.discreteBlockSize_.y,
            handle.discreteBlockSize_.z);
    printf ("   size3_: %d %d %d\n", handle.size3_);
    printf ("   size2_: %d %d %d\n", handle.size2_);
    printf ("   size1_: %d %d %d\n", handle.size1_);
    printf ("   w2h3_: %f\n", handle.w2h3_);
    printf ("End print from device\n");
    printf ("--------------------------------------------------------------\n");
}

//=================================================================
