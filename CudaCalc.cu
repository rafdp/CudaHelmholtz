#include "hip/hip_runtime.h"

//=================================================================

#include "CudaCalc.h"

__global__ void PrintGrid (complex_t* data, int size)
{
    printf ("Printing grid 2d (%X)\n", data);
    for (int i = 0; i < size; i++)
    { 
	printf ("  ");
        for (int j = 0; j < size; j++)
	{
            printf ("(%.2e %.2e) ", data[i*size + j].real (), data[i*size + j].imag ());
	}
	printf ("\n");
    }
    printf ("PrintGrid2D ended (%X)\n", data);
}

__global__ void PrintGrid3 (complex_t* data, int size)
{
    printf ("Printing grid 3d (%X)\n", data);
    for (int i = 0; i < size; i++)
    {
	printf ("Layer %d\n", i);
        for (int j = 0; j < size; j++)
	{
	    printf ("  ");
            for (int k = 0; k < size; k++)
	    {
		int idx = i*size*size + j*size + k;
		printf ("(%.2e %.2e) ", 
			data[idx].real (), 
			data[idx].imag ());
	    }
	    printf ("\n");
	}
	printf ("\n");
    }
    printf ("PrintGrid3D ended (%X)\n", data);
}

__global__ void DevicePrintData (InputDataOnDevice * inputDataPtr)
{
    /*    Point3DDevice_t<float> sourcePos_;
    float w_; //DROP
    thrust::complex<float> uiCoeff_;
    Point3DDevice_t<float> anomalyPos_;
    Point3DDevice_t<float> anomalySize_;
    Point3DDevice_t<int>   discretizationSize_;
    Point3DDevice_t<int>   discreteBlockSize_;
    int                    size3_;
    int                    size2_;
    int                    size1_;
    float                  w2h3_;*/

    printf ("device address %x\n", inputDataPtr);

    InputDataOnDevice& handle = *(inputDataPtr);

    printf ("--------------------------------------------------------------\n");
    printf ("Printing from device:\n");
    printf ("   sourcePos_: %f %f %f\n",
            inputDataPtr->sourcePos_.x,
            handle.sourcePos_.y,
            handle.sourcePos_.z);
    printf ("   uiCoeff: %f i*%f\n",
            handle.uiCoeff_.real (),
            handle.uiCoeff_.imag ());
    printf ("   anomalySize_: %f %f %f\n",
            handle.anomalySize_.x,
            handle.anomalySize_.y,
            handle.anomalySize_.z);
    printf ("   discretizationSize_: %d %d %d\n",
            handle.discretizationSize_.x,
            handle.discretizationSize_.y,
            handle.discretizationSize_.z);
    printf ("   discreteBlockSize_: %d %d %d\n",
            handle.discreteBlockSize_.x,
            handle.discreteBlockSize_.y,
            handle.discreteBlockSize_.z);
    printf ("   size3_: %d\n", handle.size3_);
    printf ("   size2_: %d\n", handle.size2_);
    printf ("   size1_: %d\n", handle.size1_);
    printf ("   w2h3_: %f\n", handle.w2h3_);
    printf ("End print from device\n");
    printf ("--------------------------------------------------------------\n");
}

__global__ void ReduceEmittersToReceiver (InputDataOnDevice * inputDataPtr,
                                          complex_t* deviceKMatrixPtr,
                                          complex_t* reductedA_solution,
                                          int* sequence,
                                          point_t* indexesPtr)
{
    int receiver = blockIdx.x * blockDim.x + threadIdx.x;
    BTransformReduceUnary tempFunctor (deviceKMatrixPtr,
                                       indexesPtr, 
                                       receiver,
                                       inputDataPtr);
    ComplexAddition complexSum;
    *(reductedA_solution + receiver) = 
    thrust::transform_reduce (thrust::device, 
                              sequence,
                              sequence + inputDataPtr->size3_,
                              tempFunctor,
                              complex_t (0.0f, 0.0f),
                              complexSum);
}

const char * cublasGetErrorString (hipblasStatus_t error)
{
    switch (error)
    {
        case HIPBLAS_STATUS_SUCCESS:
            return "HIPBLAS_STATUS_SUCCESS";

        case HIPBLAS_STATUS_NOT_INITIALIZED:
            return "HIPBLAS_STATUS_NOT_INITIALIZED";

        case HIPBLAS_STATUS_ALLOC_FAILED:
            return "HIPBLAS_STATUS_ALLOC_FAILED";

        case HIPBLAS_STATUS_INVALID_VALUE:
            return "HIPBLAS_STATUS_INVALID_VALUE";

        case HIPBLAS_STATUS_ARCH_MISMATCH:
            return "HIPBLAS_STATUS_ARCH_MISMATCH";

        case HIPBLAS_STATUS_MAPPING_ERROR:
            return "HIPBLAS_STATUS_MAPPING_ERROR";

        case HIPBLAS_STATUS_EXECUTION_FAILED:
            return "HIPBLAS_STATUS_EXECUTION_FAILED";

        case HIPBLAS_STATUS_INTERNAL_ERROR:
            return "HIPBLAS_STATUS_INTERNAL_ERROR";
    }

    return "<unknown>";
}

const char * cusolverGetErrorString (hipsolverStatus_t error)
{
    switch (error)
    {
        case HIPSOLVER_STATUS_SUCCESS:
            return "The operation completed successfully";

        case HIPSOLVER_STATUS_NOT_INITIALIZED:
            return "The library was not initialized";

        case HIPSOLVER_STATUS_INVALID_VALUE:
            return "Invalid parameters were passed";

        case HIPSOLVER_STATUS_ARCH_MISMATCH:
            return "The device only supports compute capability 2.0 and above";

        case HIPSOLVER_STATUS_INTERNAL_ERROR:
            return "HIPSOLVER_STATUS_INTERNAL_ERROR";
    }

    return "<unknown>";
}

//=================================================================
