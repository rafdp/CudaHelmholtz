#include "hip/hip_runtime.h"

//=================================================================

#include "CudaCalc.h"

__device__ InputDataOnDevice * inputDataPtr;

struct ModifyKMatrix
{
__device__
    complex_t operator() (complex_t& k, point_t& pos)
    {
        point_t dr = {inputDataPtr->sourcePos_.x - pos.x,
                      inputDataPtr->sourcePos_.y - pos.y,
                      inputDataPtr->sourcePos_.z - pos.z};
        float len = dr.len ();
        return inputDataPtr->w2h3_ * thrust::exp (inputDataPtr->uiCoeff_ * len) / (4 * 3.141592f * len) * k;
    }
};

/*
w2h3ds2ui *
exp (Gcoeff * len) / (4 * PI_ * len)
*/
struct SetAMatrix
{
    complex_t * deviceKMatrixPtr;
    point_t * deviceIndexesPtr;

    SetAMatrix (complex_t * deviceKMatrixPtr_, point_t * deviceIndexesPtr_) :
        deviceKMatrixPtr (deviceKMatrixPtr_),
        deviceIndexesPtr (deviceIndexesPtr_)
    {}

__device__
    complex_t operator() (int idx)
    {
        int idx1 = idx % inputDataPtr->size3_; // receiver
        int idx2 = idx / inputDataPtr->size3_; // emitter
        if (idx1 == idx2) return thrust::complex <float> (0.0f, 0.0f);

        point_t pos1 = *(deviceIndexesPtr + idx1);
        point_t pos2 = *(deviceIndexesPtr + idx2);
        point_t dr = {pos1.x-pos2.x,
                      pos1.y-pos2.y,
                      pos1.z-pos2.z};
        float len = dr.len ();

	
//--------------------------------------------------------------------+
// using ui in point   idx1   , maybe will need to tune               |
// if row-major order is used:                                        |
//                                  00 10 20                          |
//                                  01 11 21                          |
//                                  02 12 22                          |
//                                  03 13 23                          |
//                                  04 14 24                          |
//                                  05 15 25                          |
//                                  06 16 26                          |
//                                  07 17 27                          |
//                                  08 18 28                          |
//                                  09 19 29                          |
// every column contains all the points for a single receiver         |
// when converting to column-major:                                   |
// sequential receiver storage                                        |
//                                  00 01 02 03 04 05 06 07 08 09     |
//                                  10 11 12 13 14 15 16 17 18 19     |
//                                  20 21 22 23 24 25 26 27 28 29 ... |
//--------------------------------------------------------------------+


        return (*(deviceKMatrixPtr + idx2))* thrust::exp (inputDataPtr->uiCoeff_ * len) / (4 * 3.141592f * len);
    }
};

//Aii = - ui
struct ModifyAMatrix
{
    complex_t * deviceAMatrixPtr;
    point_t * deviceIndexesPtr;

    ModifyAMatrix (complex_t * deviceAMatrixPtr_, point_t * deviceIndexesPtr_) :
        deviceAMatrixPtr (deviceAMatrixPtr_),
        deviceIndexesPtr (deviceIndexesPtr_)
    {}

__device__
    void operator() (int idx)
    {
        point_t pos = *(deviceIndexesPtr + idx);
        point_t dr = {inputDataPtr->sourcePos_.x - pos.x,
                      inputDataPtr->sourcePos_.y - pos.y,
                      inputDataPtr->sourcePos_.z - pos.z};
        float len = dr.len ();
        if (len < 0.0000001 && len > 0.0000001) return;
        *(deviceAMatrixPtr + idx*(inputDataPtr->size3_+1)) = 
		-thrust::exp (inputDataPtr->uiCoeff_ * len) / (4 * 3.141592f * len);

    }
};

struct QLReduction
{
	const point_t receiver;
    complex_t * deviceLambdaPtr;
    point_t * deviceIndexesPtr;
    complex_t * deviceKMatrixPtr;

    __host__
	QLReduction (point_t receiver_,
                 complex_t * deviceLambdaPtr_,
                 point_t * deviceIndexesPtr_,
                 complex_t * deviceKMatrixPtr_) :
        receiver (receiver_),
        deviceLambdaPtr (deviceLambdaPtr_),
        deviceIndexesPtr (deviceIndexesPtr_),
        deviceKMatrixPtr (deviceKMatrixPtr_)
    {}

    __device__
	complex_t operator()(int idx) const
	{
		point_t& r = *(deviceIndexesPtr + idx);

		point_t dr = {r.x - receiver.x +
                                      inputDataPtr->discreteBlockSize_.x / 2.0,
                                      r.y - receiver.y +
                                      inputDataPtr->discreteBlockSize_.y / 2.0,
                                      r.z - receiver.z +
                                      inputDataPtr->discreteBlockSize_.z / 2.0};

		float len = dr.len ();

        	if (len < 0.0000001 && len > 0.0000001) return complex_t (0.0f, 0.0f);
		return (*(deviceKMatrixPtr + idx)) * (complex_t (1.0f, 0.0f) + *(deviceLambdaPtr + idx)) * 
                thrust::exp(inputDataPtr -> uiCoeff_ * len) / (4 * 3.141592f * len);
	}
};

struct IndexFromSequence
{
    __device__
    point_t operator() (int idx) const
    {

        point_t point = { 1.0f * (idx % inputDataPtr->size1_),
                                         1.0f * ((idx / inputDataPtr->size1_) % inputDataPtr->discretizationSize_.y),
                                         1.0f * (idx / inputDataPtr->size2_)};
        point = {(float) (point.x*inputDataPtr->discreteBlockSize_.x*1.0f +
                 inputDataPtr->anomalyPos_.x +
                 inputDataPtr->discreteBlockSize_.x / 2.0),
                 (float) (point.y*inputDataPtr->discreteBlockSize_.y*1.0f +
                 inputDataPtr->anomalyPos_.y +
                 inputDataPtr->discreteBlockSize_.y / 2.0),
                 (float) (point.z*inputDataPtr->discreteBlockSize_.z*1.0f +
                 inputDataPtr->anomalyPos_.z +
                 inputDataPtr->discreteBlockSize_.z / 2.0)};
        return point;
    }
};

struct MatVecFunctor : MatVecFunctorBase
{
    hipblasHandle_t cublasH;
    hipComplex* device_A_;
    size_t size_;
    

    __host__
    MatVecFunctor (hipblasHandle_t cH,
                   complex_t * deviceAMatrixPtr,
                   size_t size) :
        cublasH   (cH),
        device_A_ (reinterpret_cast<hipComplex*> (deviceAMatrixPtr)),
        size_     (size)
    {}

    __host__
    void operator()(hipComplex* source, hipComplex* destination) 
    {
        hipComplex one = {1.0f, 0.0f};
        hipComplex zero = {0.0f, 0.0f};
        hipblasCgemv (cublasH, HIPBLAS_OP_N, size_, size_, &one,
                    device_A_, size_, source, 1, &zero, destination, 1);
    }
};

#define CENTER_INDEX (2*size.y+1)*size.x
struct FillRadialQ_lq
{
    point_t * deviceIndexesPtr;
    complex_t* Q_lq;
    pointInt_t size;
    int l;
    int q;
    
    __host__
    FillRadialQ_lq (point_t * deviceIndexesPtr_,
                    complex_t * Q_lq_,
                    pointInt_t size_,
                    int l_,
                    int q_) :
    deviceIndexesPtr (deviceIndexesPtr_),
    Q_lq             (Q_lq_),
    size             (size_),
    l                (l_),
    q                (q_)
    {}
    
    __device__
    void operator()(int idx) const
    {
	if (idx < 2*size.x) *(Q_lq + idx) = complex_t (0.0f, 0.0f);
	if (idx < 2*size.y) *(Q_lq + idx*2*size.x) = complex_t (0.0f, 0.0f);
	
        int idxy = idx % size.x;
        int idxx = idx / size.x;
        
        int emIdx = l*size.x*size.y;
        int recIdx = q*size.x*size.y + idxy * size.x + idxx;
        
       	//printf ("!!!!!!!!!!Q_lq fill, got idx %d, x %d, y %d, em %d, rec %d\n", idx, idxx, idxy, emIdx, recIdx);
        point_t em = *(deviceIndexesPtr + emIdx);
        if (emIdx == recIdx)
        {
            *(Q_lq + CENTER_INDEX)  = complex_t (0.0f, 0.0f); 
	    return;
        }
        point_t rec = *(deviceIndexesPtr + recIdx);
        
        point_t dr = {rec.x - em.x,
            rec.y - em.y,
            rec.z - em.z};
        
        float len = dr.len ();
	complex_t fill_val = (inputDataPtr->w2h3_ * thrust::exp (inputDataPtr->uiCoeff_ * len) / (4 * (3.141592f) * len));    
        *(Q_lq + CENTER_INDEX + (size.x*2)*idxy + idxx) =
        *(Q_lq + CENTER_INDEX - (size.x*2)*idxy + idxx) =
        *(Q_lq + CENTER_INDEX + (size.x*2)*idxy - idxx) =
        *(Q_lq + CENTER_INDEX - (size.x*2)*idxy - idxx) = fill_val;
        
    }
        
};

struct FillRadialQ
{
    point_t * deviceIndexesPtr;
    complex_t* Q_full;
    pointInt_t size;
    
    __host__
    FillRadialQ (point_t * deviceIndexesPtr_,
                 complex_t * Q_full_,
                 pointInt_t size_) :
    deviceIndexesPtr (deviceIndexesPtr_),
    Q_full           (Q_full_),
    size             (size_)
    {}
    
    __device__
    void operator()(int idx) const
    {
	int idxx = idx % size.x;
	idx /= size.x;
	int idxy = idx % size.y;
	idx /= size.y;
	int q = idx % size.z;
	int l = idx / size.z;

        int gridSize = size.x*size.y*4;
	if (idxy == 0) *(Q_full + (l*size.z + q) * gridSize + idxx) = complex_t (0.0f, 0.0f);
	if (idxx == 0) *(Q_full + (l*size.z + q) * gridSize + idxy * 2*size.x) = complex_t (0.0f, 0.0f);
        
        int emIdx = l*size.x*size.y;
        int recIdx = q*size.x*size.y + idxy * size.x + idxx;
        
        if (emIdx == recIdx)
        {
            *(Q_full + (l*size.z + q) * gridSize + CENTER_INDEX)  = complex_t (0.0f, 0.0f); 
	    return;
        }
        point_t em = *(deviceIndexesPtr + emIdx);
        point_t rec = *(deviceIndexesPtr + recIdx);
        
        point_t dr = {rec.x - em.x,
                      rec.y - em.y,
                      rec.z - em.z};
        
        float len = dr.len ();
	complex_t fill_val = (inputDataPtr->w2h3_ * thrust::exp (inputDataPtr->uiCoeff_ * len) / (4 * (3.141592f) * len));    
        *(Q_full + (l*size.z + q) * gridSize + CENTER_INDEX + (size.x*2)*idxy + idxx) =
        *(Q_full + (l*size.z + q) * gridSize + CENTER_INDEX - (size.x*2)*idxy + idxx) =
        *(Q_full + (l*size.z + q) * gridSize + CENTER_INDEX + (size.x*2)*idxy - idxx) =
        *(Q_full + (l*size.z + q) * gridSize + CENTER_INDEX - (size.x*2)*idxy - idxx) = fill_val;
        
    }
        
};

struct FillV
{
    point_t * deviceIndexesPtr;
    complex_t* source;
    complex_t* V;
    pointInt_t size;
    
    __host__
    FillV (point_t * deviceIndexesPtr_,
	   complex_t * source_,
           complex_t * V_,
           pointInt_t size_) :
        deviceIndexesPtr (deviceIndexesPtr_),
        source (source_),
        V      (V_),
        size   (size_)
    {}

    __device__
    void operator()(int idx) 
    {
        int idxOld = idx;
        int idxx = idx % size.x;
	idx /= size.x;
        int idxy = idx % size.y;
	int idxz = idx / size.y;
        idx *= size.x; 
        int recIdx = idxz*size.x*size.y + idxy * size.x + idxx;
	
	point_t rec = *(deviceIndexesPtr + recIdx);
        point_t dr = {inputDataPtr->sourcePos_.x - rec.x,
                      inputDataPtr->sourcePos_.y - rec.y,
     		      inputDataPtr->sourcePos_.z - rec.z};
        float len = dr.len ();
	complex_t value = thrust::exp (inputDataPtr->uiCoeff_ * len) / (4 * 3.141592f * len); 
	*(V + idxz*4*size.x*size.y + CENTER_INDEX + (size.x*2)*idxy + idxx) =
            *(source + idxOld) * value;
	
	if (idxy)         *(V + idxz*4*size.x*size.y + CENTER_INDEX - (size.x*2)*idxy + idxx) = complex_t (0.0f, 0.0f);
	if (idxx)         *(V + idxz*4*size.x*size.y + CENTER_INDEX + (size.x*2)*idxy - idxx) = complex_t (0.0f, 0.0f);
	if (idxx && idxy) *(V + idxz*4*size.x*size.y + CENTER_INDEX - (size.x*2)*idxy - idxx) = complex_t (0.0f, 0.0f);
        if (!idxy)
	{
	          *(V + idxz*4*size.x*size.y + idxx) = complex_t (0.0f, 0.0f);
	          *(V + idxz*4*size.x*size.y + idxx + size.x) = complex_t (0.0f, 0.0f);
	          *(V + idxz*4*size.x*size.y + idxx*size.x*2 + size.x*2*size.y) = complex_t (0.0f, 0.0f);
        if (idxx) *(V + idxz*4*size.x*size.y + idxx*size.x*2) = complex_t (0.0f, 0.0f);
	}
    }
};
/*
struct FillV_q
{
    point_t * deviceIndexesPtr;
    complex_t* source;
    complex_t* V_q;
    pointInt_t size;
    int q;
    
    __host__
    FillV_q (point_t * deviceIndexesPtr_,
	     complex_t * source_,
             complex_t * V_q_,
             pointInt_t size_,
             int q_) :
        deviceIndexesPtr (deviceIndexesPtr_),
        source (source_),
        V_q    (V_q_),
        size   (size_),
        q      (q_)
    {}

    __device__
    void operator()(int idx) const
    {
        int idxx = idx % size.x;
        int idxy = idx / size.x;
        
        int recIdx = q*size.x*size.y + idxy * size.x + idxx;
	
	point_t rec = *(deviceIndexesPtr + recIdx);
        point_t dr = {inputDataPtr->sourcePos_.x - rec.x,
                      inputDataPtr->sourcePos_.y - rec.y,
     		      inputDataPtr->sourcePos_.z - rec.z};
        float len = dr.len ();
	complex_t value = thrust::exp (inputDataPtr->uiCoeff_ * len) / (4 * 3.141592f * len); 
         
	*(V_q + CENTER_INDEX + (size.x*2)*idxy + idxx) =
                       *(source + q*size.x*size.y + idxy * size.x + idxx) * value;
        *(V_q + CENTER_INDEX - 2*size.x - (size.x*2)*idxy + idxx)     = complex_t (0.0f, 0.0f);
	*(V_q + CENTER_INDEX +            (size.x*2)*idxy - idxx - 1) = complex_t (0.0f, 0.0f);
        *(V_q + CENTER_INDEX - 2*size.x - (size.x*2)*idxy - idxx - 1) = complex_t (0.0f, 0.0f);
    }
};
*/
const int OP_ADD = '+',
          OP_SUB = '-',
	  OP_MUL = '*';

struct ElementwiseOperation
{
    complex_t* modifiable;
    complex_t* source;
    int op;
    __host__
    ElementwiseOperation (complex_t * mod_,
                           complex_t * source_,
			   int op_) :
        modifiable (mod_),
        source     (source_),
	op         (op_)
    {}

    __device__
    void operator()(int idx) const
    {
        if (op == OP_ADD) *(modifiable + idx) += *(source + idx);
	else
        if (op == OP_SUB) *(modifiable + idx) -= *(source + idx);
	else
        if (op == OP_MUL) *(modifiable + idx) *= *(source + idx);
    }
};

struct MatrixElementwiseVectorMultiplication
{
    complex_t* proxy;
    complex_t* Q_full;
    complex_t* V_full;
    pointInt_t size;
    __host__
    MatrixElementwiseVectorMultiplication (complex_t* proxy_,
		                           complex_t* Q_full_,
		                           complex_t* V_full_,
					   pointInt_t size_) : 
        proxy  (proxy_),
	Q_full (Q_full_),
	V_full (V_full_),
	size   (size_)
    {}

    __device__
    void operator () (int idx)
    {
	int idxV = idx % (size.x*size.y*4*size.z);
   
        *(proxy + idx) = *(Q_full + idx) * *(V_full + idxV);

    }    

};

struct FillS_l
{
    complex_t* destination;
    complex_t* acc;
    pointInt_t size;
    int l;
    
    __host__
    FillS_l (complex_t * destination_,
             complex_t * acc_,
             pointInt_t size_,
             int l_) :
        destination (destination_),
        acc (acc_),
        size     (size_),
        l (l_)
    {}

    __device__
    void operator()(int idx) const
    {
        int idxy = idx % size.x;
        int idxx = idx / size.x;
        *(destination + l*size.x*size.y + idxy*size.x + idxx) = 
        *(acc + (size.x*2)*idxy + idxx) / (4.0f*size.x*size.y);
    }
};


struct MinusSourceUi
{
    point_t* deviceIndexesPtr;
    complex_t* source;
    complex_t* destination;
    __host__ 
    MinusSourceUi (point_t* deviceIndexesPtr_,
		   complex_t* source_,
		   complex_t* destination_) : 
        deviceIndexesPtr (deviceIndexesPtr_),
	source           (source_),
	destination      (destination_)
    {}

    __device__ 
    void operator () (int idx)
    { 
	point_t rec = *(deviceIndexesPtr + idx);
        point_t dr = {inputDataPtr->sourcePos_.x - rec.x,
                      inputDataPtr->sourcePos_.y - rec.y,
     		      inputDataPtr->sourcePos_.z - rec.z};
        float len = dr.len ();
	complex_t value =  *(source + idx) * thrust::exp (inputDataPtr->uiCoeff_ * len) / (4 * 3.141592f * len);
        //printf ("subtracting %d %.2e %.2e\n", idx, (source+idx)->real(), (destination+idx)->real ());	
	//printf ("before: %.2e, after: %.e2\n", (destination+idx)->real (), (*(destination + idx) - value).real ());
	*(destination + idx) -= value;
    }

};

#undef CENTER_INDEX

#define CF(val) \
if ((cufft_error = val) != HIPFFT_SUCCESS) \
printf ("ERROR on line %d, code %d\n", __LINE__, cufft_error);

struct MatVecFunctorFFT : MatVecFunctorBase
{
    complex_t* deviceDS2MatrixPtr;
    point_t * deviceIndexesPtr;
    int* seq;
    pointInt_t size;
    thrust::device_vector <complex_t> Q_full;
    thrust::device_vector <complex_t> V_full;
    thrust::device_vector <complex_t> result_proxy;
    thrust::device_vector <complex_t> accumulator;
    hipfftHandle planQ;
    hipfftHandle planV;
    int cufft_error;
    __host__
    MatVecFunctorFFT (complex_t * deviceDS2MatrixPtr_,
                      point_t *   deviceIndexesPtr_,
                      int* seq_,
                      pointInt_t size_) :
        deviceDS2MatrixPtr (deviceDS2MatrixPtr_),
        deviceIndexesPtr   (deviceIndexesPtr_),
        seq                (seq_),
        size               (size_),
	Q_full             (4*size.x*size.y*size.z*size.z, complex_t (0.0f, 0.0f)),
	V_full             (4*size.x*size.y*size.z,        complex_t (0.0f, 0.0f)),
	result_proxy       (4*size.x*size.y*size.z*size.z, complex_t (0.0f, 0.0f)),
	accumulator        (4*size.x*size.y*size.z,        complex_t (0.0f, 0.0f)),
	planQ              (),
	planV              (),
	cufft_error        (HIPFFT_SUCCESS)
    {        
	int sizes[2] = {size.x*2, size.y*2};
        int gridSize = 4*size.x*size.y;
        CF(hipfftPlanMany (&planQ, 2, sizes, nullptr, 1, gridSize, nullptr, 1, gridSize, HIPFFT_C2C, size.z*size.z))
        CF(hipfftPlanMany (&planV, 2, sizes, nullptr, 1, gridSize, nullptr, 1, gridSize, HIPFFT_C2C, size.z))
        FillRadialQ fr (deviceIndexesPtr, Q_full.data ().get (), size);
        thrust::for_each (thrust::device, seq, seq + size.x*size.y*size.z*size.z, fr);
        CF(hipfftExecC2C(planQ, 
                        reinterpret_cast<hipfftComplex*> (Q_full.data ().get ()),
                        reinterpret_cast<hipfftComplex*> (Q_full.data ().get ()), HIPFFT_FORWARD))

    }

    __host__
    ~MatVecFunctorFFT ()
    {
        CF (hipfftDestroy (planQ))
	CF (hipfftDestroy (planV))
    }

    __host__
    void operator()(hipComplex* source, hipComplex* destination)
    {
        const int gridSize = 2*size.x*2*size.y;

#define TIME_PROFILE(x)
	

        TIME_PROFILE(timespec ts[20] = {};)
	TIME_PROFILE(int ts_index = 0;)
        TIME_PROFILE(hipDeviceSynchronize ();)
        TIME_PROFILE(clock_gettime(CLOCK_REALTIME, ts + ts_index); ts_index++;)//
	FillV fillV (deviceIndexesPtr, 
		     reinterpret_cast<complex_t*> (source),
		     V_full.data ().get (),
		     size);
	
	thrust::for_each (thrust::device, seq, seq + size.x*size.y*size.z, fillV);
        
        TIME_PROFILE(hipDeviceSynchronize ();)
        TIME_PROFILE(clock_gettime(CLOCK_REALTIME, ts + ts_index); ts_index++;)//
	CF(hipfftExecC2C(planV, 
                        reinterpret_cast<hipfftComplex*> (V_full.data ().get ()),
                        reinterpret_cast<hipfftComplex*> (V_full.data ().get ()), HIPFFT_FORWARD))
	

        TIME_PROFILE(hipDeviceSynchronize ();)
        TIME_PROFILE(clock_gettime(CLOCK_REALTIME, ts + ts_index); ts_index++;)//
	MatrixElementwiseVectorMultiplication mevm (result_proxy.data ().get (),
			                            Q_full.data ().get (), 
						    V_full.data ().get (), 
						    size);

        thrust::for_each (thrust::device, seq, seq + 4*size.x*size.y*size.z*size.z, mevm);
        //PrintGrid<<<1, 1>>> (Q_full.data ().get (), size.x*2);


        TIME_PROFILE(hipDeviceSynchronize ();)
        TIME_PROFILE(clock_gettime(CLOCK_REALTIME, ts + ts_index); ts_index++;)//
	accumulator.assign (gridSize*size.z, complex_t (0.0f, 0.0f));
	//hipDeviceSynchronize ();
	for (int l = 0; l < size.z; l++)
        {
            for (int q = 0; q < size.z; q++)
            {   
		ElementwiseOperation lsf (accumulator.data ().get () + l*gridSize, 
		                          result_proxy.data ().get () + (l*size.z+q)*gridSize, OP_ADD);
                thrust::for_each (thrust::device, seq, seq + gridSize, lsf);
            }
        } 
        hipDeviceSynchronize ();
	
	TIME_PROFILE(hipDeviceSynchronize ();)
	TIME_PROFILE(clock_gettime(CLOCK_REALTIME, ts + ts_index); ts_index++;)//
        // 
        CF(hipfftExecC2C(planV, 
                        reinterpret_cast<hipfftComplex*> (accumulator.data ().get ()),
                        reinterpret_cast<hipfftComplex*> (accumulator.data ().get ()), HIPFFT_BACKWARD))
        
        TIME_PROFILE(hipDeviceSynchronize ();)
        TIME_PROFILE(clock_gettime(CLOCK_REALTIME, ts + ts_index); ts_index++;)//
	
	
	for (int l = 0; l < size.z; l++)
	{
            FillS_l fs (reinterpret_cast<complex_t*> (destination),
                        accumulator.data().get() + l*gridSize, size, l);
            
            thrust::for_each (thrust::device, seq, seq + size.x*size.y, fs);
        }

        TIME_PROFILE(hipDeviceSynchronize ();)
        TIME_PROFILE(clock_gettime(CLOCK_REALTIME, ts + ts_index); ts_index++;)//
	
	ElementwiseOperation ds2_mul (reinterpret_cast<complex_t*> (destination), deviceDS2MatrixPtr, OP_MUL);
	thrust::for_each (thrust::device, seq, seq + size.x*size.y*size.z, ds2_mul);
        
        TIME_PROFILE(hipDeviceSynchronize ();)
        TIME_PROFILE(clock_gettime(CLOCK_REALTIME, ts + ts_index); ts_index++;)//
        MinusSourceUi msu (deviceIndexesPtr, 
			  reinterpret_cast<complex_t*> (source),
			  reinterpret_cast<complex_t*> (destination));
        thrust::for_each (thrust::device, seq, seq + size.x*size.y*size.z, msu);
        TIME_PROFILE(hipDeviceSynchronize ();)
        TIME_PROFILE(clock_gettime(CLOCK_REALTIME, ts + ts_index); ts_index++;)//
        
	TIME_PROFILE(for (int i = 1; i < ts_index; i++)
	    printf ("%d->%d took %f mks\n", i - 1, i, 
	            ((ts[i].tv_sec - ts[i-1].tv_sec)*1000000000.0f + ts[i].tv_nsec - ts[i-1].tv_nsec)/1000.0f);)
    }
};

#undef CF

extern "C"
void ExternalKernelCaller (InputData_t* inputDataPtr_, std::vector<std::complex<float> >* retData)
{
    InputData_t& inputData = *inputDataPtr_;

    InputDataOnDevice* deviceInputData = nullptr;

    hipblasStatus_t cublas_status = HIPBLAS_STATUS_SUCCESS;
    hipError_t cudaStat = hipSuccess;

    CC(hipMalloc ((void**) &deviceInputData, sizeof (InputDataOnDevice)));

    CC(hipMemcpyToSymbol(HIP_SYMBOL(inputDataPtr), &deviceInputData, sizeof(InputDataOnDevice*)));

    int size3 = inputData.discretizationSize_[0] *
                inputData.discretizationSize_[1] *
                inputData.discretizationSize_[2];

    InputDataOnDevice hostDataCopy = {inputData.sourcePos_,
                                      complex_t (0.0f, (float) (2*3.141592f*inputData.f_/inputData.c_)),
                                      inputData.anomalyPos_,
                                      inputData.anomalySize_,
                                      inputData.discretizationSize_,
                                      inputData.discreteBlockSize_,
                                      size3,
                                      inputData.discretizationSize_[0] *
                                      inputData.discretizationSize_[1],
                                      inputData.discretizationSize_[0],
                                      (float)(4*3.141592f*3.141592f*inputData.f_*inputData.f_*
                                      inputData.discreteBlockSize_[0]*
				      inputData.discreteBlockSize_[1]*
				      inputData.discreteBlockSize_[2])};

    CC(hipMemcpy (deviceInputData, &hostDataCopy, sizeof (InputDataOnDevice), hipMemcpyHostToDevice));
    
    printf ("About to call kernel\n");
    DevicePrintData<<<1, 1>>> (deviceInputData);
    CC(hipDeviceSynchronize ());
    printf ("Kernel returned\n");

    thrust::host_vector<complex_t > hostDs2Matrix (size3);

    for (int x = 0; x < inputData.discretizationSize_[0]; x++)
    {
        for (int y = 0; y < inputData.discretizationSize_[1]; y++)
        {
            for (int z = 0; z < inputData.discretizationSize_[2]; z++)
            {
                int currentIndex = (x + y*inputData.discretizationSize_[0] + z*inputData.discretizationSize_[0]*inputData.discretizationSize_[1]);
                hostDs2Matrix[currentIndex] = complex_t (float (inputData.ds2_[currentIndex]), 0.0);
            }
        }
    }

    thrust::device_vector<complex_t> deviceKMatrix   (hostDs2Matrix);
    thrust::device_vector<complex_t> deviceDS2Matrix (deviceKMatrix);
    
    thrust::device_vector<point_t > indexes (size3);
    
    thrust::tabulate (indexes.begin(), indexes.end(), IndexFromSequence ());
    
    thrust::transform (deviceKMatrix.begin (), deviceKMatrix.end (), indexes.begin (), deviceKMatrix.begin (), ModifyKMatrix ());
    
    timespec ts00 = {}, ts01 = {};
    clock_gettime(CLOCK_REALTIME, &ts00); // Works on Linux
    thrust::device_vector<complex_t > deviceAMatrix (size3*size3);
    
    SetAMatrix sMatrixSetter (deviceKMatrix.data ().get (), indexes.data ().get ());

    thrust::tabulate (deviceAMatrix.begin (), deviceAMatrix.end (), sMatrixSetter);


    /// ////////////////////////////////////
    /// solution part (linear system, not fft)
    /// ////////////////////////////////////


    /// 1. Creating handles

    hipblasHandle_t cublasH = nullptr;
    CB(hipblasCreate(&cublasH));

    //hipsolverHandle_t cudenseH = nullptr;
    //CS(hipsolverDnCreate(&cudenseH));
    
    /// 2. Setting up data

    thrust::device_vector<complex_t> ones (size3, complex_t (-1.0f, 0.0f)); // is it -1 or -1 - i ?
    thrust::device_vector<complex_t> reductedA_solution (size3, 0.0f);

    complex_t alpha (1.0f, 0.0f);
    complex_t beta (0.0f, 0.0f);
    
    thrust::device_vector<int> seq (size3 * size3);
    thrust::sequence (seq.begin (), seq.end ());
    
    ReduceEmittersToReceiver 
    <<<inputData.discretizationSize_[0]*
       inputData.discretizationSize_[1], 
       inputData.discretizationSize_[2]>>> 
        (deviceInputData,
         deviceKMatrix.data ().get (),
         reductedA_solution.data ().get (),
         seq.data().get (),
         indexes.data ().get ());
    



    /// need to subtract ui from every diagonal element of A
    /// strategy1: run tabulate on something of size size3 and modify A alongside
    /// strategy2: run for_each on a sequence, but need to create sequence of size size3

    ///using strategy2
    
    ModifyAMatrix modificatorA (deviceAMatrix.data ().get (), indexes.data ().get ());
    thrust::for_each (seq.begin(), seq.begin() + size3, modificatorA);
    clock_gettime(CLOCK_REALTIME, &ts01);


    /// 3. Querying workspace for cusolverDn

    /*int workspaceSize = 0;

    CS(hipsolverDnCgeqrf_bufferSize(cudenseH,
                                   size3,
                                   size3,
                                   reinterpret_cast <hipComplex*> (deviceAMatrix.data ().get ()),
                                   size3,
                                   &workspaceSize));

    thrust::device_vector<complex_t> workspace (workspaceSize);


    /// 4. Computing QR decomposition

    thrust::device_vector<complex_t> tau (size3);

    CC(hipMalloc ((void**)&devInfo, sizeof(int)));


    CS(hipsolverDnCgeqrf(cudenseH,
                        size3,
                        size3,
                        reinterpret_cast <hipComplex*> (deviceAMatrix.data ().get ()),
                        size3,
                        reinterpret_cast <hipComplex*> (tau.data ().get ()),
                        reinterpret_cast <hipComplex*> (workspace.data ().get ()),
                        workspaceSize,
                        devInfo));
    CC(hipDeviceSynchronize());


    /// 5. compute Q^H*B
    CS(hipsolverDnCunmqr(cudenseH,
                        HIPBLAS_SIDE_LEFT,
                        HIPBLAS_OP_C,
                        size3,
                        1,
                        size3, //k 	host 	input 	number of elementary relfections
                        reinterpret_cast <hipComplex*> (deviceAMatrix.data ().get ()),
                        size3,
                        reinterpret_cast <hipComplex*> (tau.data ().get ()),
                        reinterpret_cast <hipComplex*> (reductedA_solution.data ().get ()),
                        size3,
                        reinterpret_cast <hipComplex*> (workspace.data ().get ()),
                        workspaceSize,
                        devInfo));




    CC(hipDeviceSynchronize());

    
    /// 6. solve Rx = Q^H*B
    CB(hipblasCtrsm(cublasH,
                   HIPBLAS_SIDE_LEFT,
                   HIPBLAS_FILL_MODE_UPPER,
                   HIPBLAS_OP_N,
                   HIPBLAS_DIAG_NON_UNIT,
                   size3,
                   1,
                   reinterpret_cast <hipComplex*> (&alpha),
                   reinterpret_cast <hipComplex*> (deviceAMatrix.data ().get ()),
                   size3,
                   reinterpret_cast <hipComplex*> (reductedA_solution.data ().get ()),
                   size3));
    //3-6. Bicgstab solution*/

    thrust::host_vector <complex_t> x_0 (size3, complex_t (1.0f, 0.0f));
    thrust::device_vector <complex_t> x (x_0);
    thrust::device_vector <complex_t> t0 (x_0);
    thrust::device_vector <complex_t> t1 (x_0);

	
    MatVecFunctor matvecf_ (cublasH, deviceAMatrix.data().get (), size3);
    timespec ts10 = {}, ts11 = {};
    clock_gettime(CLOCK_REALTIME, &ts10); // Works on Linux
    MatVecFunctorFFT matvecf (deviceDS2Matrix.data().get (), 
		              indexes.data (). get (), 
			      seq.data ().get (), 
			      inputData.discretizationSize_);
    
    clock_gettime(CLOCK_REALTIME, &ts11); // Works on Linux
    alpha = complex_t (-1.0f, 0.0f);
    //printf ("About to print A matrix\n");
    //PrintGrid <<<1, 1>>> (deviceAMatrix.data().get(), inputData.discretizationSize_[0]*inputData.discretizationSize_[1]*inputData.discretizationSize_[2]);
    
    matvecf_ (reinterpret_cast<hipComplex*> (x.data().get ()),
             reinterpret_cast<hipComplex*> (t1.data().get ()));
    
    for (int i = 0; i < 1; i++)
    matvecf (reinterpret_cast<hipComplex*> (x.data().get ()),
              reinterpret_cast<hipComplex*> (t0.data().get ()));
    
    /*matvecf (reinterpret_cast<hipComplex*> (x.data().get ()),
              reinterpret_cast<hipComplex*> (t0.data().get ()));
    matvecf (reinterpret_cast<hipComplex*> (x.data().get ()),
              reinterpret_cast<hipComplex*> (t0.data().get ()));*/
    float norm0 = 0.0f;
    float norm1 = 0.0f;
    
    CB (hipblasScnrm2 (cublasH, size3, reinterpret_cast<hipComplex*> (t1.data().get ()), 1, &norm1));
    
    CB (hipblasScnrm2 (cublasH, size3, reinterpret_cast<hipComplex*> (t0.data().get ()), 1, &norm0));

    //printf ("FFT result:\n");
    //PrintGrid3 <<<1, 1>>> (t0.data().get (), inputData.discretizationSize_[0]);   
    //hipDeviceSynchronize (); 
    //printf ("Matvec result:\n");
    //PrintGrid3 <<<1, 1>>> (t1.data().get (), inputData.discretizationSize_[0]);    
    //hipDeviceSynchronize ();
    CB (hipblasCaxpy(cublasH, size3, reinterpret_cast<hipComplex*> (&alpha), 
                    reinterpret_cast<hipComplex*> (t0.data().get ()), 1, 
                    reinterpret_cast<hipComplex*> (t1.data().get ()), 1));
    
    float norm = 0.0f;
    
    CB (hipblasScnrm2 (cublasH, size3, reinterpret_cast<hipComplex*> (t1.data().get ()), 1, &norm));
    
    printf ("Got norm (difference) = %e\nnorm FFT = %*e\nnorm Matvec = %*e\n", norm, 13, norm0, 10, norm1);
    //return;
  //--------------------------------------------------------------------------------------------------------  
  //--------------------------------------------------------------------------------------------------------  
  //--------------------------------------------------------------------------------------------------------  
  //--------------------------------------------------------------------------------------------------------  
  // /*
    BiCGStabCudaSolver solver (size3, reductedA_solution.data().get (), x.data().get ());

    timespec ts0 = {}, ts1 = {};
    clock_gettime(CLOCK_REALTIME, &ts0); // Works on Linux
    solver.solve (&matvecf_);
    clock_gettime(CLOCK_REALTIME, &ts1);
    unsigned long long time0 = (ts01.tv_sec - ts00.tv_sec)*1000000000 + ts01.tv_nsec-ts00.tv_nsec; 
    unsigned long long time1 = (ts1.tv_sec - ts0.tv_sec)*1000000000 + ts1.tv_nsec-ts0.tv_nsec; 
    printf ("MATRIX took %f ms\n", (time1 + time0)/1000000.0f);
    ts0 = {};
    ts1 = {};
    clock_gettime(CLOCK_REALTIME, &ts0); // Works on Linux
    solver.solve (&matvecf);
    clock_gettime(CLOCK_REALTIME, &ts1);
    hipDeviceSynchronize ();
    unsigned long long time2 = (ts11.tv_sec - ts10.tv_sec)*1000000000 + ts11.tv_nsec-ts10.tv_nsec; 
    unsigned long long time3 = (ts1.tv_sec - ts0.tv_sec)*1000000000 + ts1.tv_nsec-ts0.tv_nsec; 
    printf ("FFT took %f ms\n", (time2+time3)/1000000.0f);

    printf ("Matrix is %.5f times faster than FFT (%d^3)\n", (1.0*time2 + time3)/(time1 + time0), inputData.discretizationSize_[0]);


    CC(hipDeviceSynchronize());

    CB (hipblasCcopy (cublasH, size3,
                         (reinterpret_cast <hipComplex*> (x.data().get ())), 1,
                         (reinterpret_cast <hipComplex*> (reductedA_solution.data().get ())), 1));
    
    alpha = complex_t (1.0f, 0.0f);
    
    CB(hipblasCscal(cublasH, size3,
                    reinterpret_cast <hipComplex*> (&alpha),
                    reinterpret_cast <hipComplex*> (reductedA_solution.data ().get ()), 1));





    /// 7. receiver convolution

    for (int i = 0; i < inputData.Nreceivers_; i++)
    {
        QLReduction qlRed (inputData.receivers_[i], 
			   reductedA_solution.data().get(), 
			   indexes.data ().get (), deviceKMatrix.data ().get ());
        complex_t init (0.0f, 0.0f);
        ComplexAddition complexSum;
        thrust::transform (seq.begin (), seq.begin () + size3, ones.begin(), qlRed);
        (*retData)[i] = thrust::reduce (ones.begin(), ones.end(), init, complexSum);
    }


    CB(hipblasDestroy (cublasH));
    CC(hipFree (deviceInputData));
    printf ("Cuda part ended\n");
//*/

}


//=================================================================
