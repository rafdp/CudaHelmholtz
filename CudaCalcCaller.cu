#include "hip/hip_runtime.h"

//=================================================================

#include <thrust/device_vector.h>
#include <thrust/transform.h>
#include <thrust/sequence.h>
#include <thrust/copy.h>
#include <thrust/fill.h>
#include <thrust/replace.h>
#include <thrust/transform_reduce.h>
#include <thrust/memory.h>
#include <thrust/complex.h>
#include <thrust/device_new.h>
#include <thrust/functional.h>
#include "hipblas.h"


#include "CudaCalc.h"

struct InputDataOnDevice
{
    Point3DDevice_t<float> sourcePos_;
    float w_; //DROP
    thrust::complex<float> uiCoeff_;
    Point3DDevice_t<float> anomalyPos_;
    Point3DDevice_t<float> anomalySize_;
    Point3DDevice_t<int>   discretizationSize_;
    Point3DDevice_t<int>   discreteBlockSize_;
    int                    size3_;
    int                    size2_;
    int                    size1_;
    float                  w2h3_;
};


__global__ void BornForRecieversKernel (int * P_recv, InputData_t* INPUT_DATA_PTR);

__global__ void DevicePrintData (InputDataOnDevice * inputDataPtr);

__global__ void DevicePrint ();


template <typename T>
__host__ __device__
float Point3DDevice_t<T>::len ()
{
    return sqrtf (x*x + y*y + z*z);
}

__device__ thrust::complex<float> * deviceKMatrixPtr;
__device__ Point3DDevice_t<float> * deviceIndexesPtr;
__device__ InputDataOnDevice * inputDataPtr;

struct ModifyKMatrix
{
__device__
    thrust::complex<float> operator() (Point3DDevice_t<float>& pos, thrust::complex<float>& k)
    {
        Point3DDevice_t<float> dr = {inputDataPtr->sourcePos_.x - pos.x,
                                     inputDataPtr->sourcePos_.y - pos.y,
                                     inputDataPtr->sourcePos_.z - pos.z};
        float len = dr.len ();
        return k*inputDataPtr->w2h3_ * thrust::exp (inputDataPtr->uiCoeff_ * len) / (4 * 3.141592f * len);
    }
};

/*
w2h3ds2ui *
exp (Gcoeff * len) / (4 * PI_ * len)
*/
struct SetAMatrix
{
__device__
    thrust::complex<float> operator() (int idx)
    {
        int idx1 = idx % inputDataPtr->size3_;
        int idx2 = idx / inputDataPtr->size3_;
        Point3DDevice_t<float> pos1 = *(deviceIndexesPtr + idx1);
        Point3DDevice_t<float> pos2 = *(deviceIndexesPtr + idx2);
        Point3DDevice_t<float> dr = {pos1.x-pos2.x,
                                     pos1.y-pos2.y,
                                     pos1.z-pos2.z};
        float len = dr.len ();

//--------------------------------------------------------------------+
// using ui in point   idx2   , maybe will need to tune               |
// if row-major order is used:                                        |
//                                  00 10 20                          |
//                                  01 11 21                          |
//                                  02 12 22                          |
//                                  03 13 23                          |
//                                  04 14 24                          |
//                                  05 15 25                          |
//                                  06 16 26                          |
//                                  07 17 27                          |
//                                  08 18 28                          |
//                                  09 19 29                          |
// every column contains all the points for a single receiver         |
// when converting to column-major:                                   |
// row = receiver                                                     |
//                                  00 01 02 03 04 05 06 07 08 09     |
//                                  10 11 12 13 14 15 16 17 18 19     |
//                                  20 21 22 23 24 25 26 27 28 29 ... |
//--------------------------------------------------------------------+


        return (*(deviceKMatrixPtr + idx2)) * thrust::exp (inputDataPtr->uiCoeff_ * len) / (4 * 3.141592f * len);
    }
};


struct IndexFromSequence
{
    __device__
    Point3DDevice_t<float> operator() (int idx) const
    {
        Point3DDevice_t<float> point = { 1.0f * (idx % inputDataPtr->size2_),
                                         1.0f * ((idx / inputDataPtr->size1_) % inputDataPtr->discretizationSize_.y),
                                         1.0f * (idx / inputDataPtr->size2_)};
        point = {(float) (point.x*inputDataPtr->discreteBlockSize_.x*1.0f +
                 inputDataPtr->anomalyPos_.x +
                 inputDataPtr->discreteBlockSize_.x / 2.0),
                 (float) (point.y*inputDataPtr->discreteBlockSize_.y*1.0f +
                 inputDataPtr->anomalyPos_.y +
                 inputDataPtr->discreteBlockSize_.y / 2.0),
                 (float) (point.z*inputDataPtr->discreteBlockSize_.z*1.0f +
                 inputDataPtr->anomalyPos_.z +
                 inputDataPtr->discreteBlockSize_.z / 2.0)};
        return point;
    }
};


extern "C"
void ExternalKernelCaller (InputData_t* inputDataPtr_)
{
	InputData_t& inputData = *inputDataPtr_;

	hipMalloc ((void**) &inputDataPtr, sizeof (InputDataOnDevice));

	InputDataOnDevice* hostInputData = nullptr;

	hipMemcpy (&hostInputData, &inputDataPtr, sizeof (InputDataOnDevice*), hipMemcpyDeviceToHost);

    int size3 = inputData.discretizationSize_[0] *
                inputData.discretizationSize_[1] *
                inputData.discretizationSize_[2];

    #define PointConversion(var, type)\
    (Point3DDevice_t<type>) \
    {(type)(inputData.var.x),  \
     (type)(inputData.var.y), \
     (type)(inputData.var.z)}

    InputDataOnDevice hostDataCopy = {PointConversion (sourcePos_, float),
                                      (float) (2*3.141592f*inputData.f_),
                                      thrust::complex<float> (0, (float) (2*3.141592f*inputData.f_/inputData.c_)),
                                      PointConversion (anomalyPos_, float),
                                      PointConversion (anomalySize_, float),
                                      (Point3DDevice_t<int>){inputData.discretizationSize_[0],
                                       inputData.discretizationSize_[1],
                                       inputData.discretizationSize_[2]},
                                      (Point3DDevice_t<int>){inputData.discreteBlockSize_[0],
                                       inputData.discreteBlockSize_[1],
                                       inputData.discreteBlockSize_[2]},
                                      size3,
                                      inputData.discretizationSize_[0] *
                                      inputData.discretizationSize_[1],
                                      inputData.discretizationSize_[0],
                                      (float)(4*3.141592f*3.141592f*inputData.f_*inputData.f_*
                                      inputData.discreteBlockSize_[0]*inputData.discreteBlockSize_[1]*inputData.discreteBlockSize_[2])};

    #undef PointConversion

    hipMemcpy (hostInputData, &hostDataCopy, sizeof (InputDataOnDevice), hipMemcpyHostToDevice);

    printf ("About to call kernel\n");
    DevicePrintData<<<1, 1>>> (hostInputData);
    hipDeviceSynchronize ();
    printf ("Kernel returned\n");

    /*hipblasHandle_t handle = 0;
    hipblasCreate(&handle);

    thrust::host_vector<float> hostDs2Matrix (size3);

    for (int x = 0; x < inputData->discretizationSize_[0]; x++)
    {
        for (int y = 0; y < inputData->discretizationSize_[1]; y++)
        {
            for (int z = 0; z < inputData->discretizationSize_[2]; z++)
            {
                int currentIndex = (x + y*inputData->discretizationSize_[0] + z*inputData->discretizationSize_[0]*inputData->discretizationSize_[1]);
                hostDs2Matrix[currentIndex] = thrust::complex<float> (1.0, 1.0) * inputData.ds2[currentIndex];
            }
        }
    }

    thrust::device_vector<thrust::complex<float> > deviceKMatrix (hostDs2Matrix);

    deviceKMatrixPtr = thrust::raw_pointer_cast(deviceKMatrix.data ());

    thrust::device_vector<Point3DDevice_t<float> > indexes (size3);

    deviceIndexesPtr = thrust::raw_pointer_cast(indexes.data ());

    thrust::tabulate (indexes.begin(), indexes.end(), IndexFromSequence ());

    thrust::transform (indexes.begin (), indexes.end (), deviceKMatrix.begin (), deviceKMatrix.begin (), ModifyKMatrix ());

    thrust::device_vector<thrust::complex<float> > deviceAMatrix (size3*size3);

    thrust::tabulate (deviceAMatrix.begin (), deviceAMatrix.end (), SetAMatrix ());*/




}


//=================================================================
