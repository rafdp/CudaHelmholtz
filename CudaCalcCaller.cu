
//=================================================================

#include "CudaCalc.h"
#define FFT_FUNCTOR
//#define MATRIX_FUNCTOR
//#define QR_SOLUTION 
#define TIME_TESTING

#ifdef TIME_TESTING
#define TT(x) x
#else
#define TT(x) 
#endif

#include "MatVecAlgorithms.cu"
extern "C"
void ExternalKernelCaller (InputData_t* inputDataPtr_, std::vector<std::complex<float> >* retData)
{ 
    TT(timespec initStart = {};)
    TT(timespec initEnd = {};)
    TT(timespec algEnd = {};)

    TT(clock_gettime(CLOCK_REALTIME, &initStart);)
    InputData_t& inputData = *inputDataPtr_;

    hipblasStatus_t cublas_status = HIPBLAS_STATUS_SUCCESS;

    int size3 = inputData.discretizationSize_[0] *
                inputData.discretizationSize_[1] *
                inputData.discretizationSize_[2];
    complex_t uiCoeff = complex_t (0.0f, (float) (2*3.141592f*inputData.f_/inputData.c_));
    float w2h3 = 4*3.141592f*3.141592f*inputData.f_*inputData.f_*
                                       inputData.discreteBlockSize_[0]*
		 		       inputData.discreteBlockSize_[1]*
				       inputData.discreteBlockSize_[2];



    thrust::host_vector<complex_t > hostDs2Matrix (inputData.ds2_, inputData.ds2_ + size3);
    thrust::device_vector<complex_t> deviceKMatrix   (hostDs2Matrix);
    thrust::device_vector<complex_t> deviceDS2Matrix (hostDs2Matrix);
    
    thrust::device_vector<point_t > indexes (size3);
    IndexFromSequence index_filler (inputData.discretizationSize_,
		                    inputData.discreteBlockSize_,
				    inputData.anomalyPos_); 
    thrust::tabulate (indexes.begin(), indexes.end(), index_filler);
    
    thrust::device_vector<int> seq (size3 * 4*inputData.discretizationSize_[2]);
    thrust::sequence (seq.begin (), seq.end ());

    ModifyKMatrix k_matrix_modifier (inputData.sourcePos_, uiCoeff, w2h3);
    thrust::transform (deviceKMatrix.begin (), 
		       deviceKMatrix.end (), 
		       indexes.begin (), 
		       deviceKMatrix.begin (), 
		       k_matrix_modifier);

    thrust::device_vector <complex_t> reductedA_solution (size3, 0.0f);
    hipblasHandle_t cublasH = nullptr;
    CB(hipblasCreate(&cublasH));
#ifdef FFT_FUNCTOR


    thrust::device_vector <complex_t> x                 (size3, complex_t (1.0f, 0.0f));

    
    ReduceEmittersToReceiver 
    <<<inputData.discretizationSize_[0]*
       inputData.discretizationSize_[1], 
       inputData.discretizationSize_[2]>>> 
        (deviceKMatrix.data ().get (),
         reductedA_solution.data ().get (),
         seq.data().get (),
         indexes.data ().get (),
	 uiCoeff,
	 size3);

    MatVecFunctorFFT matvecf (deviceDS2Matrix.data().get (), 
		              indexes.data (). get (), 
			      seq.data ().get (), 
			      inputData.discretizationSize_,
			      w2h3,
			      uiCoeff,
			      inputData.sourcePos_);
    BiCGStabCudaSolver solver (size3, reductedA_solution.data().get (), x.data().get ());
    
    TT(clock_gettime(CLOCK_REALTIME, &initEnd);)
    size_t usedSize = solver.solve (&matvecf);

    TT(clock_gettime(CLOCK_REALTIME, &algEnd);)
    CB (hipblasCcopy (cublasH, size3,
                     (reinterpret_cast <hipComplex*> (x.data().get ())), 1,
                     (reinterpret_cast <hipComplex*> (reductedA_solution.data().get ())), 1));
    hipDeviceSynchronize ();
#endif 
#ifdef MATRIX_FUNCTOR

    
    thrust::device_vector<complex_t > deviceAMatrix (size3*size3);
    
    SetAMatrix aMatrixSetter (deviceKMatrix.data ().get (), 
                              indexes.data ().get (), 
			      inputData.sourcePos_, 
			      uiCoeff,
			      size3);

    thrust::tabulate (deviceAMatrix.begin (), deviceAMatrix.end (), aMatrixSetter);
    ReduceEmittersToReceiver 
    <<<inputData.discretizationSize_[0]*
       inputData.discretizationSize_[1], 
       inputData.discretizationSize_[2]>>> 
        (deviceKMatrix.data ().get (),
         reductedA_solution.data ().get (),
         seq.data().get (),
         indexes.data ().get (),
	 uiCoeff,
	 size3);
    
    ModifyAMatrix modificatorA (deviceAMatrix.data ().get (), 
                                indexes.data ().get (),
                                inputData.sourcePos_,
                                size3,
                                uiCoeff);
    
    PrintGrid<<<1, 1>>> (deviceAMatrix.data().get (), 4);
    thrust::device_vector <complex_t> x (size3, complex_t (1.0f, 0.0f));
    
    MatVecFunctor matvecf (cublasH, deviceAMatrix.data().get (), size3);
    hipDeviceSynchronize ();
    BiCGStabCudaSolver solver (size3, reductedA_solution.data().get (), x.data().get ());

    TT(clock_gettime(CLOCK_REALTIME, &initEnd);)
    size_t usedSize = solver.solve (&matvecf);
    TT(clock_gettime(CLOCK_REALTIME, &algEnd);)
    CB (hipblasCcopy (cublasH, size3,
                     (reinterpret_cast <hipComplex*> (x.data().get ())), 1,
                     (reinterpret_cast <hipComplex*> (reductedA_solution.data().get ())), 1));
    hipDeviceSynchronize ();
#endif
#ifdef QR_SOLUTION

    hipError_t cudaStat = hipSuccess;
    hipsolverStatus_t cusolver_status = HIPSOLVER_STATUS_SUCCESS;
    int* devInfo = nullptr;
    int devInfoHost = 0;
    hipsolverHandle_t cudenseH = nullptr;
    CS(hipsolverDnCreate(&cudenseH));
    
    thrust::device_vector<complex_t > deviceAMatrix (size3*size3);
    
    SetAMatrix aMatrixSetter (deviceKMatrix.data ().get (), 
                              indexes.data ().get (), 
			      inputData.sourcePos_, 
			      uiCoeff,
			      size3);

    thrust::tabulate (deviceAMatrix.begin (), deviceAMatrix.end (), aMatrixSetter);
    ReduceEmittersToReceiver 
    <<<inputData.discretizationSize_[0]*
       inputData.discretizationSize_[1], 
       inputData.discretizationSize_[2]>>> 
        (deviceKMatrix.data ().get (),
         reductedA_solution.data ().get (),
         seq.data().get (),
         indexes.data ().get (),
	 uiCoeff,
	 size3);

    ModifyAMatrix modificatorA (deviceAMatrix.data ().get (), 
                                indexes.data ().get (),
                                inputData.sourcePos_,
                                size3,
                                uiCoeff);
    thrust::for_each (seq.begin(), seq.begin() + size3, modificatorA);
    int workspaceSize = 0;

    CS(hipsolverDnCgeqrf_bufferSize(cudenseH,
                                   size3,
                                   size3,
                                   reinterpret_cast <hipComplex*> (deviceAMatrix.data ().get ()),
                                   size3,
                                   &workspaceSize));

    thrust::device_vector<complex_t> workspace (workspaceSize);


    TT(clock_gettime(CLOCK_REALTIME, &initEnd);)
    /// 4. Computing QR decomposition

    thrust::device_vector<complex_t> tau (size3);

    CC(hipMalloc ((void**)&devInfo, sizeof(int)));


    CS(hipsolverDnCgeqrf(cudenseH,
                        size3,
                        size3,
                        reinterpret_cast <hipComplex*> (deviceAMatrix.data ().get ()),
                        size3,
                        reinterpret_cast <hipComplex*> (tau.data ().get ()),
                        reinterpret_cast <hipComplex*> (workspace.data ().get ()),
                        workspaceSize,
                        devInfo));


    /// 5. compute Q^H*B
    CS(hipsolverDnCunmqr(cudenseH,
                        HIPBLAS_SIDE_LEFT,
                        HIPBLAS_OP_C,
                        size3,
                        1,
                        size3, //k 	host 	input 	number of elementary relfections
                        reinterpret_cast <hipComplex*> (deviceAMatrix.data ().get ()),
                        size3,
                        reinterpret_cast <hipComplex*> (tau.data ().get ()),
                        reinterpret_cast <hipComplex*> (reductedA_solution.data ().get ()),
                        size3,
                        reinterpret_cast <hipComplex*> (workspace.data ().get ()),
                        workspaceSize,
                        devInfo));

    
    complex_t complex_one (1.0f, 0.0f); 
    /// 6. solve Rx = Q^H*B
    CB(hipblasCtrsm(cublasH,
                   HIPBLAS_SIDE_LEFT,
                   HIPBLAS_FILL_MODE_UPPER,
                   HIPBLAS_OP_N,
                   HIPBLAS_DIAG_NON_UNIT,
                   size3,
                   1,
                   reinterpret_cast <hipComplex*> (&complex_one),
                   reinterpret_cast <hipComplex*> (deviceAMatrix.data ().get ()),
                   size3,
                   reinterpret_cast <hipComplex*> (reductedA_solution.data ().get ()),
                   size3));
    TT(clock_gettime(CLOCK_REALTIME, &algEnd);)
    
    size_t free_byte = 0;
    size_t total_byte = 0;

    hipMemGetInfo( &free_byte, &total_byte );
    size_t usedSize = total_byte-free_byte;
    CC(hipFree (devInfo));
#endif

#ifdef TIME_TESTING
    unsigned long long timeInit = (initEnd.tv_sec - initStart.tv_sec)*1000000000 + initEnd.tv_nsec-initStart.tv_nsec; 
    unsigned long long timeAlg =  (algEnd.tv_sec - initEnd.tv_sec)*1000000000 +    algEnd.tv_nsec-initEnd.tv_nsec; 

#ifdef FFT_FUNCTOR
#define FILE_ "fft_"
#elif defined MATRIX_FUNCTOR
#define FILE_ "matrix_"
#elif defined QR_SOLUTION
#define FILE_ "qr_"
#endif

#define WRITE_FILE(suffix, data, coeff) \
    FILE* fft_##suffix = fopen (FILE_ #suffix ".txt", "a"); \
    if (!fft_##suffix) return; \
    float value##suffix = (data); \
    fprintf (fft_##suffix, "%d %f\n", inputData.discretizationSize_[0], value##suffix/coeff); \
    fclose (fft_##suffix);
    WRITE_FILE (time_init, timeInit, 1e9f)
    WRITE_FILE (time_alg, timeAlg, 1e9f)
    WRITE_FILE (time_full, timeInit + timeAlg, 1e9f)
    WRITE_FILE (size, usedSize, (1024.0f*1024.0f))

#undef WRITE_FILE
#undef FILE_

#else
    thrust::device_vector<complex_t> ones (size3, complex_t (-1.0f, 0.0f)); 
    for (int i = 0; i < inputData.Nreceivers_; i++)
    {
        QLReduction qlRed (inputData.receivers_[i], 
			   reductedA_solution.data().get(), 
			   indexes.data ().get (), 
			   deviceKMatrix.data ().get (),
			   inputData.discreteBlockSize_,
			   uiCoeff);
        complex_t init (0.0f, 0.0f);
        ComplexAddition complexSum;
        thrust::transform (seq.begin (), seq.begin () + size3, ones.begin(), qlRed);
        (*retData)[i] = thrust::reduce (ones.begin(), ones.end(), init, complexSum);
    }

    CB(hipblasDestroy (cublasH));
#ifdef QR_SOLUTION
    CS(hipsolverDnDestroy (cudenseH));
#endif
#endif

    printf ("Cuda part ended\n");
    /// ////////////////////////////////////
    /// solution part (linear system, not fft)
    /// ////////////////////////////////////

    /// 1. Creating handles
    /*hipblasHandle_t cublasH = nullptr;
    CB(hipblasCreate(&cublasH));
#ifdef QR_SOLUTION
    hipsolverStatus_t cusolver_status = HIPSOLVER_STATUS_SUCCESS;
    int* devInfo = nullptr;
    int devInfoHost = 0;
    hipsolverHandle_t cudenseH = nullptr;
    CS(hipsolverDnCreate(&cudenseH));
#endif

 /// 2. Setting up data
	    
    thrust::device_vector<complex_t> reductedA_solution (size3, 0.0f);
    ReduceEmittersToReceiver 
    <<<inputData.discretizationSize_[0]*
       inputData.discretizationSize_[1], 
       inputData.discretizationSize_[2]>>> 
        (deviceKMatrix.data ().get (),
         reductedA_solution.data ().get (),
         seq.data().get (),
         indexes.data ().get (),
	 uiCoeff,
	 size3);
    

    //begin comment here

    /// need to subtract ui from every diagonal element of A
    /// strategy1: run tabulate on something of size size3 and modify A alongside
    /// strategy2: run for_each on a sequence, but need to create sequence of size size3

    ///using strategy2
#ifndef FFT_FUNCTOR    
    ModifyAMatrix modificatorA (deviceAMatrix.data ().get (), 
                                indexes.data ().get (),
                                inputData.sourcePos_,
                                size3,
                                uiCoeff);
    thrust::for_each (seq.begin(), seq.begin() + size3, modificatorA);
    clock_gettime(CLOCK_REALTIME, &ts01);

    //end comment here


    /// 3. Querying workspace for cusolverDn
#ifdef QR_SOLUTION
    int workspaceSize = 0;

    CS(hipsolverDnCgeqrf_bufferSize(cudenseH,
                                   size3,
                                   size3,
                                   reinterpret_cast <hipComplex*> (deviceAMatrix.data ().get ()),
                                   size3,
                                   &workspaceSize));

    thrust::device_vector<complex_t> workspace (workspaceSize);


    /// 4. Computing QR decomposition

    thrust::device_vector<complex_t> tau (size3);

    CC(hipMalloc ((void**)&devInfo, sizeof(int)));


    CS(hipsolverDnCgeqrf(cudenseH,
                        size3,
                        size3,
                        reinterpret_cast <hipComplex*> (deviceAMatrix.data ().get ()),
                        size3,
                        reinterpret_cast <hipComplex*> (tau.data ().get ()),
                        reinterpret_cast <hipComplex*> (workspace.data ().get ()),
                        workspaceSize,
                        devInfo));
    CC(hipDeviceSynchronize());


    /// 5. compute Q^H*B
    CS(hipsolverDnCunmqr(cudenseH,
                        HIPBLAS_SIDE_LEFT,
                        HIPBLAS_OP_C,
                        size3,
                        1,
                        size3, //k 	host 	input 	number of elementary relfections
                        reinterpret_cast <hipComplex*> (deviceAMatrix.data ().get ()),
                        size3,
                        reinterpret_cast <hipComplex*> (tau.data ().get ()),
                        reinterpret_cast <hipComplex*> (reductedA_solution.data ().get ()),
                        size3,
                        reinterpret_cast <hipComplex*> (workspace.data ().get ()),
                        workspaceSize,
                        devInfo));

    CC(hipDeviceSynchronize());

    
    complex_t complex_one (1.0f, 0.0f); 
    /// 6. solve Rx = Q^H*B
    CB(hipblasCtrsm(cublasH,
                   HIPBLAS_SIDE_LEFT,
                   HIPBLAS_FILL_MODE_UPPER,
                   HIPBLAS_OP_N,
                   HIPBLAS_DIAG_NON_UNIT,
                   size3,
                   1,
                   reinterpret_cast <hipComplex*> (&complex_one),
                   reinterpret_cast <hipComplex*> (deviceAMatrix.data ().get ()),
                   size3,
                   reinterpret_cast <hipComplex*> (reductedA_solution.data ().get ()),
                   size3));
#endif
    //3-6. Bicgstab solution

    thrust::host_vector <complex_t> x_0 (size3, complex_t (1.0f, 0.0f));
    thrust::device_vector <complex_t> x (x_0);
    thrust::device_vector <complex_t> t0 (x_0);
    thrust::device_vector <complex_t> t1 (x_0);

//!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!
#ifdef MATRIX_FUNCTOR    
    MatVecFunctor matvecf (cublasH, deviceAMatrix.data().get (), size3);
#elif defined FFT_FUNCTOR
    timespec ts10 = {}, ts11 = {};
    clock_gettime(CLOCK_REALTIME, &ts10); // Works on Linux
    MatVecFunctorFFT matvecf (deviceDS2Matrix.data().get (), 
		              indexes.data (). get (), 
			      seq.data ().get (), 
			      inputData.discretizationSize_,
			      w2h3,
			      uiCoeff,
			      inputData.sourcePos_);
    
    clock_gettime(CLOCK_REALTIME, &ts11); // Works on Linux
#endif
//!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!
#ifndef QR_SOLUTION

    BiCGStabCudaSolver solver (size3, reductedA_solution.data().get (), x.data().get ());

    timespec ts0 = {}, ts1 = {};
    /*clock_gettime(CLOCK_REALTIME, &ts0); // Works on Linux
    size_t usedSize = solver.solve (&matvecf_);
    clock_gettime(CLOCK_REALTIME, &ts1);
    unsigned long long time0 = (ts01.tv_sec - ts00.tv_sec)*1000000000 + ts01.tv_nsec-ts00.tv_nsec; 
    unsigned long long time1 = (ts1.tv_sec - ts0.tv_sec)*1000000000 + ts1.tv_nsec-ts0.tv_nsec; 
    printf ("MATRIX took %d %f ms\n", size1, (time1 + time0)/1000000.0f);
    FILE* matrix_data = fopen ("matrix_time.txt", "a");
    if (!matrix_data) return;
    fprintf (matrix_data, "%d %f\n", size1, (time1 + time0)/1000000000.0f);
    fclose (matrix_data);
    FILE* matrix_size = fopen ("matrix_size.txt", "a");
    if (!matrix_size) return;
    fprintf (matrix_size, "%d %f\n", size1, usedSize/1048576.0f);*
    ts0 = {};
    ts1 = {};
    clock_gettime(CLOCK_REALTIME, &ts0); // Works on Linux
    size_t usedSize = solver.solve (&matvecf);
    clock_gettime(CLOCK_REALTIME, &ts1);
    hipDeviceSynchronize ();
    unsigned long long time2 = (ts11.tv_sec - ts10.tv_sec)*1000000000 + ts11.tv_nsec-ts10.tv_nsec; 
    unsigned long long time3 = (ts1.tv_sec - ts0.tv_sec)*1000000000 + ts1.tv_nsec-ts0.tv_nsec; 
    printf ("FFT took %f ms\n", (time2+time3)/1000000.0f);

    /*FILE* fft_data = fopen ("fft_time.txt", "a");
    if (!fft_data) return;
    fprintf (fft_data, "%d %f\n", size1, (time2 + time3)/1000000000.0f);
    fclose (fft_data);
    FILE* fft_size = fopen ("fft_size.txt", "a");
    if (!fft_size) return;
    fprintf (fft_size, "%d %f\n", size1, usedSize/1048576.0f);
    //printf ("Matrix is %.5f times faster than FFT (%d^3)\n", (1.0*time2 + time3)/(time1 + time0), inputData.discretizationSize_[0]);


    CC(hipDeviceSynchronize());*

    CB (hipblasCcopy (cublasH, size3,
                         (reinterpret_cast <hipComplex*> (x.data().get ())), 1,
                         (reinterpret_cast <hipComplex*> (reductedA_solution.data().get ())), 1));
    
#endif
    /// 7. receiver convolution

    thrust::device_vector<complex_t> ones (size3, complex_t (-1.0f, 0.0f)); 
    for (int i = 0; i < inputData.Nreceivers_; i++)
    {
        QLReduction qlRed (inputData.receivers_[i], 
			   reductedA_solution.data().get(), 
			   indexes.data ().get (), 
			   deviceKMatrix.data ().get (),
			   inputData.discreteBlockSize_,
			   uiCoeff);
        complex_t init (0.0f, 0.0f);
        ComplexAddition complexSum;
        thrust::transform (seq.begin (), seq.begin () + size3, ones.begin(), qlRed);
        (*retData)[i] = thrust::reduce (ones.begin(), ones.end(), init, complexSum);
    }

    CB(hipblasDestroy (cublasH));
    printf ("Cuda part ended\n");

*/
}
#undef TT

//=================================================================
