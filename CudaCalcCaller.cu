#include "hip/hip_runtime.h"

//=================================================================

#include "CudaCalc.h"

__device__ InputDataOnDevice * inputDataPtr;

struct ModifyKMatrix
{
__device__
    complex_t operator() (complex_t& k, point_t& pos)
    {
        point_t dr = {inputDataPtr->sourcePos_.x - pos.x,
                      inputDataPtr->sourcePos_.y - pos.y,
                      inputDataPtr->sourcePos_.z - pos.z};
        float len = dr.len ();
        return inputDataPtr->w2h3_ * thrust::exp (inputDataPtr->uiCoeff_ * len) / (4 * 3.141592f * len) * k;
    }
};

/*
w2h3ds2ui *
exp (Gcoeff * len) / (4 * PI_ * len)
*/
struct SetAMatrix
{
    complex_t * deviceKMatrixPtr;
    point_t * deviceIndexesPtr;

    SetAMatrix (complex_t * deviceKMatrixPtr_, point_t * deviceIndexesPtr_) :
        deviceKMatrixPtr (deviceKMatrixPtr_),
        deviceIndexesPtr (deviceIndexesPtr_)
    {}

__device__
    complex_t operator() (int idx)
    {
        int idx1 = idx % inputDataPtr->size3_; // receiver
        int idx2 = idx / inputDataPtr->size3_; // emitter
        if (idx1 == idx2) return thrust::complex <float> (0.0f, 0.0f);

        point_t pos1 = *(deviceIndexesPtr + idx1);
        point_t pos2 = *(deviceIndexesPtr + idx2);
        point_t dr = {pos1.x-pos2.x,
                      pos1.y-pos2.y,
                      pos1.z-pos2.z};
        float len = dr.len ();

//--------------------------------------------------------------------+
// using ui in point   idx1   , maybe will need to tune               |
// if row-major order is used:                                        |
//                                  00 10 20                          |
//                                  01 11 21                          |
//                                  02 12 22                          |
//                                  03 13 23                          |
//                                  04 14 24                          |
//                                  05 15 25                          |
//                                  06 16 26                          |
//                                  07 17 27                          |
//                                  08 18 28                          |
//                                  09 19 29                          |
// every column contains all the points for a single receiver         |
// when converting to column-major:                                   |
// sequential receiver storage                                        |
//                                  00 01 02 03 04 05 06 07 08 09     |
//                                  10 11 12 13 14 15 16 17 18 19     |
//                                  20 21 22 23 24 25 26 27 28 29 ... |
//--------------------------------------------------------------------+


        return (*(deviceKMatrixPtr + idx2)) * thrust::exp (inputDataPtr->uiCoeff_ * len) / (4 * 3.141592f * len);
    }
};

//Aii = - ui
struct ModifyAMatrix
{
    complex_t * deviceAMatrixPtr;
    point_t * deviceIndexesPtr;

    ModifyAMatrix (complex_t * deviceAMatrixPtr_, point_t * deviceIndexesPtr_) :
        deviceAMatrixPtr (deviceAMatrixPtr_),
        deviceIndexesPtr (deviceIndexesPtr_)
    {}

__device__
    void operator() (int idx)
    {
        point_t pos = *(deviceIndexesPtr + idx);
        point_t dr = {inputDataPtr->sourcePos_.x - pos.x,
                      inputDataPtr->sourcePos_.y - pos.y,
                      inputDataPtr->sourcePos_.z - pos.z};
        float len = dr.len ();
        if (len < 0.0000001 && len > 0.0000001) return;
        *(deviceAMatrixPtr + idx*(inputDataPtr->size3_+1)) = 
		-thrust::exp (inputDataPtr->uiCoeff_ * len) / (4 * 3.141592f * len);

    }
};

struct QLReduction
{
	const point_t receiver;
    complex_t * deviceLambdaPtr;
    point_t * deviceIndexesPtr;
    complex_t * deviceKMatrixPtr;

    __host__
	QLReduction (point_t receiver_,
                 complex_t * deviceLambdaPtr_,
                 point_t * deviceIndexesPtr_,
                 complex_t * deviceKMatrixPtr_) :
        receiver (receiver_),
        deviceLambdaPtr (deviceLambdaPtr_),
        deviceIndexesPtr (deviceIndexesPtr_),
        deviceKMatrixPtr (deviceKMatrixPtr_)
    {}

    __device__
	complex_t operator()(int idx) const
	{
		point_t& r = *(deviceIndexesPtr + idx);

		point_t dr = {r.x - receiver.x +
                                      inputDataPtr->discreteBlockSize_.x / 2.0,
                                      r.y - receiver.y +
                                      inputDataPtr->discreteBlockSize_.y / 2.0,
                                      r.z - receiver.z +
                                      inputDataPtr->discreteBlockSize_.z / 2.0};

		float len = dr.len ();

        	if (len < 0.0000001 && len > 0.0000001) return complex_t (0.0f, 0.0f);
		return (*(deviceKMatrixPtr + idx)) * (complex_t (1.0f, 0.0f) + *(deviceLambdaPtr + idx)) * 
                thrust::exp(inputDataPtr -> uiCoeff_ * len) / (4 * 3.141592f * len);
	}
};

struct IndexFromSequence
{
    __device__
    point_t operator() (int idx) const
    {

        point_t point = { 1.0f * (idx % inputDataPtr->size1_),
                                         1.0f * ((idx / inputDataPtr->size1_) % inputDataPtr->discretizationSize_.y),
                                         1.0f * (idx / inputDataPtr->size2_)};
        point = {(float) (point.x*inputDataPtr->discreteBlockSize_.x*1.0f +
                 inputDataPtr->anomalyPos_.x +
                 inputDataPtr->discreteBlockSize_.x / 2.0),
                 (float) (point.y*inputDataPtr->discreteBlockSize_.y*1.0f +
                 inputDataPtr->anomalyPos_.y +
                 inputDataPtr->discreteBlockSize_.y / 2.0),
                 (float) (point.z*inputDataPtr->discreteBlockSize_.z*1.0f +
                 inputDataPtr->anomalyPos_.z +
                 inputDataPtr->discreteBlockSize_.z / 2.0)};
        return point;
    }
};

struct MatVecFunctor : MatVecFunctorBase
{
    hipblasHandle_t cublasH;
    hipComplex* device_A_;
    size_t size_;
    

    __host__
    MatVecFunctor (hipblasHandle_t cH,
                   complex_t * deviceAMatrixPtr,
                   size_t size) :
        cublasH   (cH),
        device_A_ (reinterpret_cast<hipComplex*> (deviceAMatrixPtr)),
        size_     (size)
    {}

    __host__
    void operator()(hipComplex* source, hipComplex* destination) const
    {
        hipComplex one = {1.0f, 0.0f};
        hipComplex zero = {0.0f, 0.0f};
        hipblasCgemv (cublasH, HIPBLAS_OP_N, size_, size_, &one,
                    device_A_, size_, source, 1, &zero, destination, 1);
    }
};
/*
struct FillRadialQ_lq
{
    complex_t* deviceKMatrixPtr;
    point_t * deviceIndexesPtr;
    complex_t* Q_lq;
    pointInt_t size;
    int l;
    int q;
    
    __host__
    FillRadialQ_lq (complex_t * deviceKMatrixPtr_,
                    point_t * deviceIndexesPtr_,
                    complex_t * Q_lq_,
                    pointInt_t size_,
                    int l_,
                    int q_) :
        deviceKMatrixPtr (deviceKMatrixPtr_),
        deviceIndexesPtr (deviceIndexesPtr_),
        Q_lq             (Q_lq_),
        size             (size_),
        l                (l_),
        q                (q_)
    {}

    __device__
    void operator()(int idx) const
    {
        int idxx = idx % (2*size.x-1);
        int idxy = idx / (2*size.x-1);
        if (idxx >= size.x) idxx -= size.x;
        else idxx = size.x - idxx - 1;
        if (idxy >= size.y) idxy -= size.y;
        else idxy = size.y - idxy - 1;
        
        int emIdx = l*size.x*size.y;
        int recIdx = q*size.x*size.y + idxx * size.x + idxy;
        
        point_t em = *(deviceIndexesPtr +emIdx);
        if (emIdx == recIdx)
        {
            point_t dr = {inputDataPtr->sourcePos_.x - em.x,
                          inputDataPtr->sourcePos_.y - em.y,
                          inputDataPtr->sourcePos_.z - em.z};
            float len = dr.len ();
            *(Q_lq + idx) = -thrust::exp (inputDataPtr->uiCoeff_ * len) / 
                            (4 * 3.141592f * len);
        }
        point_t rec = *(deviceIndexesPtr + recIdx);
        
        point_t dr = {rec.x - em.x,
            rec.y - em.y,
            rec.z - em.z};
        
        float len = dr.len ();
        
        *(Q_lq + idx) = *(deviceKMatrixPtr+emIdx) * thrust::exp (inputDataPtr->uiCoeff_ * len) / (4 * (3.141592f) * len);
        
};*/
struct FillRadialQ_lq
{
    complex_t* deviceKMatrixPtr;
    point_t * deviceIndexesPtr;
    complex_t* Q_lq;
    pointInt_t size;
    int l;
    int q;
    
    __host__
    FillRadialQ_lq (complex_t * deviceKMatrixPtr_,
                    point_t * deviceIndexesPtr_,
                    complex_t * Q_lq_,
                    pointInt_t size_,
                    int l_,
                    int q_) :
    deviceKMatrixPtr (deviceKMatrixPtr_),
    deviceIndexesPtr (deviceIndexesPtr_),
    Q_lq             (Q_lq_),
    size             (size_),
    l                (l_),
    q                (q_)
    {}
    
    __device__
    void operator()(int idx) const
    {
        int idxx = idx % size.x;
        int idxy = idx / size.x;
        
        int emIdx = l*size.x*size.y;
        int recIdx = q*size.x*size.y + idxy * size.x + idxx;
        
       	printf ("!!!!!!!!!!Q_lq fill, got idx %d, x %d, y %d, em %d, rec %d\n", idx, idxx, idxy, emIdx, recIdx);
        point_t em = *(deviceIndexesPtr + emIdx);
        int centerIdxShift = (2*size.x-1)*(size.y-1)+(size.x-1);	
        if (emIdx == recIdx)
        {
            point_t dr = {inputDataPtr->sourcePos_.x - em.x,
                          inputDataPtr->sourcePos_.y - em.y,
                          inputDataPtr->sourcePos_.z - em.z};
            float len = dr.len ();
            *(Q_lq + centerIdxShift)  = -thrust::exp (inputDataPtr->uiCoeff_ * len) /
            (4 * 3.141592f * len);
	    return;
        }
        point_t rec = *(deviceIndexesPtr + recIdx);
        
        point_t dr = {rec.x - em.x,
            rec.y - em.y,
            rec.z - em.z};
        
        float len = dr.len ();
	complex_t fill_val = (*(deviceKMatrixPtr+emIdx) * thrust::exp (inputDataPtr->uiCoeff_ * len) / (4 * (3.141592f) * len));    
        *(Q_lq + centerIdxShift + (size.x*2-1)*idxy + idxx) =
        *(Q_lq + centerIdxShift - (size.x*2-1)*idxy + idxx) =
        *(Q_lq + centerIdxShift + (size.x*2-1)*idxy - idxx) =
        *(Q_lq + centerIdxShift - (size.x*2-1)*idxy - idxx) = fill_val;
        
    }
        
};

struct FillV_q
{
    complex_t* source;
    complex_t* V_q;
    pointInt_t size;
    int q;
    
    __host__
    FillV_q (complex_t * source_,
             complex_t * V_q_,
             pointInt_t size_,
             int q_) :
        source (source_),
        V_q    (V_q_),
        size   (size_),
        q      (q_)
    {}

    __device__
    void operator()(int idx) const
    {
        int idxx = idx % size.x;
        int idxy = idx / size.x;
        
        *(V_q + (2*size.x-1)*(size.y-1)+(size.x-1)+(size.x*2-1)*idxy + idxx) =
        *(source + q*size.x*size.y + idxy * size.x + idxx);
        if (idxy) *(V_q + (2*size.x-1)*(size.y-1)+(size.x-1)-(size.x*2-1)*idxy + idxx) = complex_t (0.0f, 0.0f);
	if (idxx) *(V_q + (2*size.x-1)*(size.y-1)+(size.x-1)+(size.x*2-1)*idxy - idxx) = complex_t (0.0f, 0.0f);
        if (idxx && idxy) *(V_q + (2*size.x-1)*(size.y-1)+(size.x-1)-(size.x*2-1)*idxy - idxx) = complex_t (0.0f, 0.0f);
    }
};

struct ElementwiseMultiplierFFT
{
    complex_t* Q_lq;
    complex_t* V_q;
    __host__
    ElementwiseMultiplierFFT (complex_t * Q_lq_,
                              complex_t * V_q_) :
        Q_lq (Q_lq_),
        V_q (V_q_)
    {}

    __device__
    void operator()(int idx) const
    {
        *(Q_lq + idx) *= *(V_q + idx);
    }
};

struct LayerSumFFT
{
    complex_t* source;
    complex_t* acc;
    __host__
    LayerSumFFT (complex_t* source_,
		 complex_t* acc_) :
	source (source_),
	acc    (acc_)
    {}

    __device__
    void operator() (int idx) const
    {
	*(acc + idx) += *(source + idx);
    }
};

struct FillS_l
{
    complex_t* destination;
    complex_t* acc;
    pointInt_t size;
    int l;
    
    __host__
    FillS_l (complex_t * destination_,
             complex_t * acc_,
             pointInt_t size_,
             int l_) :
        destination (destination_),
        acc (acc_),
        size     (size_),
        l (l_)
    {}

    __device__
    void operator()(int idx) const
    {
        int idxx = idx % size.x;
        int idxy = idx / size.x;
        
        *(destination + l*size.x*size.y + idxy*size.x + idxx) = 
        *(acc + (2*size.x-1)*(size.y-1)+(size.x-1)+(size.x*2-1)*idxy + idxx)/
        ((2.0f*size.x-1)*(2.0f*size.y-1));
    }
};


struct MatVecFunctorFFT : MatVecFunctorBase
{
    complex_t* deviceKMatrixPtr;
    point_t * deviceIndexesPtr;
    int* seq;
    pointInt_t size;
    
    __host__
    MatVecFunctorFFT (complex_t * deviceKMatrixPtr_,
                      point_t *   deviceIndexesPtr_,
                      int* seq_,
                      pointInt_t size_) :
        deviceKMatrixPtr (deviceKMatrixPtr_),
        deviceIndexesPtr (deviceIndexesPtr_),
        seq      (seq_),
        size     (size_)
    {}

    __host__
    void operator()(hipComplex* source, hipComplex* destination) const
    {
        int cufft_error = HIPFFT_SUCCESS;
#define CF(val) \
if ((cufft_error = val) != HIPFFT_SUCCESS) \
printf ("ERROR on line %d, code %d\n", __LINE__, cufft_error);
        LL
        const int gridSize = (2*size.x-1)*(2*size.y-1);
	printf ("size.x = %d, size.y = %d, gridSize = %d\n", size.x, size.y, gridSize);
	thrust::device_vector <complex_t> Q_lq (gridSize,
                                                complex_t(0.0f, 0.0f));
        LL
        thrust::device_vector <complex_t> V_q (gridSize,
                                                complex_t(0.0f, 0.0f));

        LL
        thrust::device_vector <complex_t> acc (gridSize,
                                                complex_t(0.0f, 0.0f));
        hipDeviceSynchronize ();
        hipfftHandle plan;
        
        LL
	CF (hipfftCreate(&plan))
        CF (hipfftPlan2d(&plan, 2*size.x-1, 2*size.y-1, HIPFFT_C2C))
	
        LL
        printf ("About to loop over l\n");
        for (int l = 0; l < size.z; l++)
        {
        printf ("About to loop over q\n");
            for (int q = 0; q < size.z; q++)
            {   
                Q_lq.assign (gridSize, complex_t (0.0f));
                printf ("About to print Q (l=%d q=%d) before FFT\n", l, q);
                FillRadialQ_lq fr (deviceKMatrixPtr,
                                   deviceIndexesPtr,
                                   Q_lq.data ().get (),
                                   size, l, q);
		//LL 
                thrust::for_each (thrust::device, seq, seq + size.x*size.y, fr);
		//cudeDeviceSynchronize ();
		PrintGrid <<<1, 1>>> (Q_lq.data ().get(), 2*size.x-1);
                //LL
		CF(hipfftExecC2C(plan, 
                             reinterpret_cast<hipfftComplex*> (Q_lq.data ().get ()),
                             reinterpret_cast<hipfftComplex*> (Q_lq.data ().get ()), HIPFFT_FORWARD))
                //LL
		hipDeviceSynchronize ();
		printf ("About to print Q (l=%d q=%d) after FFT\n", l, q);
		hipDeviceSynchronize ();
		PrintGrid <<<1, 1>>> (Q_lq.data ().get(), 2*size.x-1);
                FillV_q fv (reinterpret_cast<complex_t*> (source),
                            V_q.data ().get (),
                            size, q);
                thrust::for_each (thrust::device, seq, seq + size.x*size.y, fv);
                //LL
                CF(hipfftExecC2C(plan, 
                             reinterpret_cast<hipfftComplex*> (V_q.data ().get ()),
                             reinterpret_cast<hipfftComplex*> (V_q.data ().get ()),
                             HIPFFT_FORWARD))
                hipDeviceSynchronize ();
		printf ("About to print V (l=%d q=%d) after FFT\n", l, q);
		hipDeviceSynchronize ();
		PrintGrid <<<1, 1>>> (V_q.data ().get(), 2*size.x-1);
                //LL
                ElementwiseMultiplierFFT ems (Q_lq.data ().get (), 
                                              V_q.data ().get ());
                thrust::for_each (thrust::device, seq, seq + gridSize, ems);
                hipDeviceSynchronize ();
		printf ("About to print Q (l=%d q=%d) after inverse elementwise Mult\n", l, q);
		hipDeviceSynchronize ();
		PrintGrid <<<1, 1>>> (Q_lq.data ().get(), 2*size.x-1);
		CF(hipfftExecC2C(plan, 
                             reinterpret_cast<hipfftComplex*> (Q_lq.data ().get ()),
                             reinterpret_cast<hipfftComplex*> (Q_lq.data ().get ()),
                             HIPFFT_BACKWARD))
		//LL
		LayerSumFFT lsf (Q_lq.data ().get (), acc.data ().get ());

		hipDeviceSynchronize ();
                printf ("About to print Q (l=%d q=%d) after inverse FFT\n", l, q);
		hipDeviceSynchronize ();
		PrintGrid <<<1, 1>>> (Q_lq.data ().get(), 2*size.x-1);
                thrust::for_each (thrust::device, seq, seq + gridSize, lsf);
		hipDeviceSynchronize ();
                printf ("About to print acc (l=%d q=%d) after accumulation\n", l, q);
		hipDeviceSynchronize ();
		PrintGrid <<<1, 1>>> (acc.data ().get(), 2*size.x-1);
		hipDeviceSynchronize ();
            }
            
            FillS_l fs (reinterpret_cast<complex_t*> (destination),
                        acc.data().get(), size, l);
            
            thrust::for_each (thrust::device, seq, seq + size.x*size.y, fs);
	    hipDeviceSynchronize ();
            printf ("About to print acc (l=%d) after full accumulation\n", l);
            hipDeviceSynchronize ();
	    PrintGrid <<<1, 1>>> (acc.data ().get(), 2*size.x-1);
	    hipDeviceSynchronize ();

	    acc.assign (gridSize, complex_t (0.0f, 0.0f));
        }
        hipfftDestroy(plan);
    }
};

extern "C"
void ExternalKernelCaller (InputData_t* inputDataPtr_, std::vector<std::complex<float> >* retData)
{
    InputData_t& inputData = *inputDataPtr_;

    InputDataOnDevice* deviceInputData = nullptr;

    hipblasStatus_t cublas_status = HIPBLAS_STATUS_SUCCESS;
    hipsolverStatus_t cusolver_status = HIPSOLVER_STATUS_SUCCESS;
    hipError_t cudaStat = hipSuccess;
    int* devInfo = nullptr;
    int devInfoHost = 0;

    CC(hipMalloc ((void**) &deviceInputData, sizeof (InputDataOnDevice)));

    CC(hipMemcpyToSymbol(HIP_SYMBOL(inputDataPtr), &deviceInputData, sizeof(InputDataOnDevice*)));

    int size3 = inputData.discretizationSize_[0] *
                inputData.discretizationSize_[1] *
                inputData.discretizationSize_[2];

    InputDataOnDevice hostDataCopy = {inputData.sourcePos_,
                                      complex_t (0.0f, (float) (2*3.141592f*inputData.f_/inputData.c_)),
                                      inputData.anomalyPos_,
                                      inputData.anomalySize_,
                                      inputData.discretizationSize_,
                                      inputData.discreteBlockSize_,
                                      size3,
                                      inputData.discretizationSize_[0] *
                                      inputData.discretizationSize_[1],
                                      inputData.discretizationSize_[0],
                                      (float)(4*3.141592f*3.141592f*inputData.f_*inputData.f_*
                                      inputData.discreteBlockSize_[0]*
				      inputData.discreteBlockSize_[1]*
				      inputData.discreteBlockSize_[2])};

    CC(hipMemcpy (deviceInputData, &hostDataCopy, sizeof (InputDataOnDevice), hipMemcpyHostToDevice));
    
    printf ("About to call kernel\n");
    DevicePrintData<<<1, 1>>> (deviceInputData);
    CC(hipDeviceSynchronize ());
    printf ("Kernel returned\n");

    thrust::host_vector<complex_t > hostDs2Matrix (size3);

    for (int x = 0; x < inputData.discretizationSize_[0]; x++)
    {
        for (int y = 0; y < inputData.discretizationSize_[1]; y++)
        {
            for (int z = 0; z < inputData.discretizationSize_[2]; z++)
            {
                int currentIndex = (x + y*inputData.discretizationSize_[0] + z*inputData.discretizationSize_[0]*inputData.discretizationSize_[1]);
                hostDs2Matrix[currentIndex] = complex_t (float (inputData.ds2_[currentIndex]), 0.0);
            }
        }
    }

    thrust::device_vector<complex_t > deviceKMatrix (hostDs2Matrix);
    
    thrust::device_vector<point_t > indexes (size3);
    
    thrust::tabulate (indexes.begin(), indexes.end(), IndexFromSequence ());
    
    thrust::transform (deviceKMatrix.begin (), deviceKMatrix.end (), indexes.begin (), deviceKMatrix.begin (), ModifyKMatrix ());
    
    thrust::device_vector<complex_t > deviceAMatrix (size3*size3);
    
    SetAMatrix sMatrixSetter (deviceKMatrix.data ().get (), indexes.data ().get ());

    thrust::tabulate (deviceAMatrix.begin (), deviceAMatrix.end (), sMatrixSetter);


    /// ////////////////////////////////////
    /// solution part (linear system, not fft)
    /// ////////////////////////////////////


    /// 1. Creating handles

    hipblasHandle_t cublasH = nullptr;
    CB(hipblasCreate(&cublasH));

    hipsolverHandle_t cudenseH = nullptr;
    CS(hipsolverDnCreate(&cudenseH));
    
    /// 2. Setting up data

    thrust::device_vector<complex_t> ones (size3, complex_t (-1.0f, 0.0f)); // is it -1 or -1 - i ?
    thrust::device_vector<complex_t> reductedA_solution (size3, 0.0f);

    complex_t alpha (1.0f, 0.0f);
    complex_t beta (0.0f, 0.0f);
    
    thrust::device_vector<int> seq (size3 * size3);
    thrust::sequence (seq.begin (), seq.end ());
    
    ReduceEmittersToReceiver 
    <<<inputData.discretizationSize_[0]*
       inputData.discretizationSize_[1], 
       inputData.discretizationSize_[2]>>> 
        (deviceInputData,
         deviceKMatrix.data ().get (),
         reductedA_solution.data ().get (),
         seq.data().get (),
         indexes.data ().get ());
    



    /// need to subtract ui from every diagonal element of A
    /// strategy1: run tabulate on something of size size3 and modify A alongside
    /// strategy2: run for_each on a sequence, but need to create sequence of size size3

    ///using strategy2
    
    ModifyAMatrix modificatorA (deviceAMatrix.data ().get (), indexes.data ().get ());
    thrust::for_each (seq.begin(), seq.begin() + size3, modificatorA);


    /// 3. Querying workspace for cusolverDn

    /*int workspaceSize = 0;

    CS(hipsolverDnCgeqrf_bufferSize(cudenseH,
                                   size3,
                                   size3,
                                   reinterpret_cast <hipComplex*> (deviceAMatrix.data ().get ()),
                                   size3,
                                   &workspaceSize));

    thrust::device_vector<complex_t> workspace (workspaceSize);


    /// 4. Computing QR decomposition

    thrust::device_vector<complex_t> tau (size3);

    CC(hipMalloc ((void**)&devInfo, sizeof(int)));


    CS(hipsolverDnCgeqrf(cudenseH,
                        size3,
                        size3,
                        reinterpret_cast <hipComplex*> (deviceAMatrix.data ().get ()),
                        size3,
                        reinterpret_cast <hipComplex*> (tau.data ().get ()),
                        reinterpret_cast <hipComplex*> (workspace.data ().get ()),
                        workspaceSize,
                        devInfo));
    CC(hipDeviceSynchronize());


    /// 5. compute Q^H*B
    CS(hipsolverDnCunmqr(cudenseH,
                        HIPBLAS_SIDE_LEFT,
                        HIPBLAS_OP_C,
                        size3,
                        1,
                        size3, //k 	host 	input 	number of elementary relfections
                        reinterpret_cast <hipComplex*> (deviceAMatrix.data ().get ()),
                        size3,
                        reinterpret_cast <hipComplex*> (tau.data ().get ()),
                        reinterpret_cast <hipComplex*> (reductedA_solution.data ().get ()),
                        size3,
                        reinterpret_cast <hipComplex*> (workspace.data ().get ()),
                        workspaceSize,
                        devInfo));




    CC(hipDeviceSynchronize());

    
    /// 6. solve Rx = Q^H*B
    CB(hipblasCtrsm(cublasH,
                   HIPBLAS_SIDE_LEFT,
                   HIPBLAS_FILL_MODE_UPPER,
                   HIPBLAS_OP_N,
                   HIPBLAS_DIAG_NON_UNIT,
                   size3,
                   1,
                   reinterpret_cast <hipComplex*> (&alpha),
                   reinterpret_cast <hipComplex*> (deviceAMatrix.data ().get ()),
                   size3,
                   reinterpret_cast <hipComplex*> (reductedA_solution.data ().get ()),
                   size3));
    //3-6. Bicgstab solution*/

    thrust::host_vector <complex_t> x_0 (size3, complex_t (1.0f, 0.0f));
    thrust::device_vector <complex_t> x (x_0);
    thrust::device_vector <complex_t> t0 (x_0);
    thrust::device_vector <complex_t> t1 (x_0);

    MatVecFunctor matvecf_ (cublasH, deviceAMatrix.data().get (), size3);
    MatVecFunctorFFT matvecf (deviceKMatrix.data().get (), indexes.data (). get (), seq.data ().get (), inputData.discretizationSize_);
    
    alpha = complex_t (-1.0f, 0.0f);
    printf ("About to print A matrix\n");
    PrintGrid3 <<<1, 1>>> (deviceAMatrix.data().get(), inputData.discretizationSize_[0]*inputData.discretizationSize_[1]);
    
    matvecf_ (reinterpret_cast<hipComplex*> (x.data().get ()),
             reinterpret_cast<hipComplex*> (t1.data().get ()));
    
    matvecf (reinterpret_cast<hipComplex*> (x.data().get ()),
              reinterpret_cast<hipComplex*> (t0.data().get ()));
    float norm0 = 0.0f;
    float norm1 = 0.0f;
    
    CB (hipblasScnrm2 (cublasH, size3, reinterpret_cast<hipComplex*> (t1.data().get ()), 1, &norm1));
    
    CB (hipblasScnrm2 (cublasH, size3, reinterpret_cast<hipComplex*> (t0.data().get ()), 1, &norm0));

    printf ("FFT result:\n");
    PrintGrid3 <<<1, 1>>> (t0.data().get (), inputData.discretizationSize_[0]);   
    hipDeviceSynchronize (); 
    printf ("Matvec result:\n");
    PrintGrid3 <<<1, 1>>> (t1.data().get (), inputData.discretizationSize_[0]);    
    hipDeviceSynchronize ();
    CB (hipblasCaxpy(cublasH, size3, reinterpret_cast<hipComplex*> (&alpha), 
                    reinterpret_cast<hipComplex*> (t0.data().get ()), 1, 
                    reinterpret_cast<hipComplex*> (t1.data().get ()), 1));
    
    float norm = 0.0f;
    
    CB (hipblasScnrm2 (cublasH, size3, reinterpret_cast<hipComplex*> (t1.data().get ()), 1, &norm));
    
    printf ("Got norm (difference) = %e\nnorm FFT = %*e\nnorm Matvec = %*e\n", norm, 13, norm0, 10, norm1);
    //return;
    
    BiCGStabCudaSolver solver (size3, reductedA_solution.data().get (), x.data().get ());

    solver.solve (&matvecf_);

    CC(hipDeviceSynchronize());

    CB (hipblasCcopy (cublasH, size3,
                         (reinterpret_cast <hipComplex*> (x.data().get ())), 1,
                         (reinterpret_cast <hipComplex*> (reductedA_solution.data().get ())), 1));
    
    alpha = complex_t (1.0f, 0.0f);
    
    CB(hipblasCscal(cublasH, size3,
                    reinterpret_cast <hipComplex*> (&alpha),
                    reinterpret_cast <hipComplex*> (reductedA_solution.data ().get ()), 1));





    /// 7. receiver convolution

    for (int i = 0; i < inputData.Nreceivers_; i++)
    {
        QLReduction qlRed (inputData.receivers_[i], reductedA_solution.data().get(), indexes.data ().get (), deviceKMatrix.data ().get ());
        complex_t init (0.0f, 0.0f);
        ComplexAddition complexSum;
        thrust::transform (seq.begin (), seq.begin () + size3, ones.begin(), qlRed);
        (*retData)[i] = thrust::reduce (ones.begin(), ones.end(), init, complexSum);
    }


    CB(hipblasDestroy (cublasH));
    CC(hipFree (deviceInputData));
    CC(hipFree (devInfo));
    
    printf ("Cuda part ended\n");


}


//=================================================================
