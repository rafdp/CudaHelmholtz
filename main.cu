#include "hip/hip_runtime.h"

#include <thrust/device_vector.h>
#include <thrust/transform.h>
#include <thrust/sequence.h>
#include <thrust/copy.h>
#include <thrust/fill.h>
#include <thrust/replace.h>
#include <thrust/complex.h>
#include <thrust/transform_reduce.h>
#include <thrust/transform.h>
#include <thrust/functional.h>
#include "Builder.h"

#define _PI 3.1415926f
__device__ thrust::complex<float> * UiPtr;
InputData_t* INPUT_DATA_PTR = nullptr;

__device__ Point3DDevice_t <float> * PointsPtr;


__device__ InputDataOnDevice * inputDataPtr;

struct BornCalculation  // rewrite inputdata use
{
	const Point3DDevice_t <float> rj;
	BornCalculation(Point3D_t _rj) : rj(_rj) {}

    __device__
	thrust::complex<float> operator()(int idx) const
	{

		InputDataOnDevice* d_inputData = inputDataPtr;
	
		Point3DDevice_t <float> r = *(PointsPtr + idx);

		Point3DDevice_t <float> dr = {r.x - rj.x, r.y - rj.y, r.z - rj.z};
						            
		return *(UiPtr + idx)  * thrust::exp(d_inputData -> uiCoeff_ * dr.len()) / (4 * _PI * dr.len());
	}
};

struct complexPlus
{
    __host__ __device__
    thrust::complex <float> operator () (const thrust::complex<float> &z1, const thrust::complex<float> &z2) const
    {
        return z1 + z2;
    }
};

struct UiMultiply
{
	__device__
	thrust::complex <float> operator()(const thrust::complex<float>& ds, const Point3DDevice_t<float>& r) const 
	{
		InputDataOnDevice* d_inputData = inputDataPtr;
		return d_inputData -> w2h3_ * ds * thrust::exp(d_inputData -> uiCoeff_ * r.len()) / (4 * _PI * r.len());
	}

};

struct IndexFromSequence
{
	__device__
		Point3DDevice_t <float> operator() (int idx) const
	{
		Point3DDevice_t<float> point = { idx % inputDataPtr->size2_,
			(idx / inputDataPtr->size1_) % inputDataPtr->discretizationSize_.y,
			idx / inputDataPtr->size2_ };
		point = { point.x*inputDataPtr->discreteBlockSize_.x*1.0f + inputDataPtr->anomalyPos_.x + inputDataPtr->discreteBlockSize_.x / 2.0,
			      point.y*inputDataPtr->discreteBlockSize_.y*1.0f + inputDataPtr->anomalyPos_.y + inputDataPtr->discreteBlockSize_.y / 2.0,
			      point.z*inputDataPtr->discreteBlockSize_.z*1.0f + inputDataPtr->anomalyPos_.z + inputDataPtr->discreteBlockSize_.z / 2.0 };
		return point;
	}
};

__global__

void DevicePrint ()
{
    printf ("--------------------------------------------------------------\n");
    printf ("threadIdx.x: %d\n", threadIdx.x);
    printf ("--------------------------------------------------------------\n");
}


__global__ void DevicePrintData (InputDataOnDevice * inputDataPtr)
{
    /*    Point3DDevice_t<float> sourcePos_;
    float w_; //DROP
    thrust::complex<float> uiCoeff_;
    Point3DDevice_t<float> anomalyPos_;
    Point3DDevice_t<float> anomalySize_;
    Point3DDevice_t<int>   discretizationSize_;
    Point3DDevice_t<int>   discreteBlockSize_;
    int                    size3_;
    int                    size2_;
    int                    size1_;
    float                  w2h3_;*/

    printf ("device address %x\n", inputDataPtr);

    InputDataOnDevice& handle = *(inputDataPtr);

    printf ("--------------------------------------------------------------\n");
    printf ("Printing from device:\n");
    printf ("   sourcePos_: %f %f %f\n",
            inputDataPtr->sourcePos_.x,
            handle.sourcePos_.y,
            handle.sourcePos_.z);
    printf ("   w: %f\n",
            handle.w_);
    printf ("   uiCoeff: %f i*%f\n",
            handle.uiCoeff_.real (),
            handle.uiCoeff_.imag ());
    printf ("   anomalySize_: %f %f %f\n",
            handle.anomalySize_.x,
            handle.anomalySize_.y,
            handle.anomalySize_.z);
    printf ("   discretizationSize_: %d %d %d\n",
            handle.discretizationSize_.x,
            handle.discretizationSize_.y,
            handle.discretizationSize_.z);
    printf ("   discreteBlockSize_: %d %d %d\n",
            handle.discreteBlockSize_.x,
            handle.discreteBlockSize_.y,
            handle.discreteBlockSize_.z);
    printf ("   size3_: %d %d %d\n", handle.size3_);
    printf ("   size2_: %d %d %d\n", handle.size2_);
    printf ("   size1_: %d %d %d\n", handle.size1_);
    printf ("   w2h3_: %f\n", handle.w2h3_);
    printf ("End print from device\n");
    printf ("--------------------------------------------------------------\n");
}

extern "C"
int main ()
{

	InputData_t inputData = {};
    //inputData.LoadData ();

	printf ("ERROR: %s\n", hipGetErrorString(hipMalloc ((void**) &deviceInputData, sizeof (InputDataOnDevice))));

    	printf ("ERROR: %s\n", hipGetErrorString(hipMemcpyToSymbol(HIP_SYMBOL(&inputData),
                                                                 &deviceInputData,
                                                                 sizeof(InputDataOnDevice*))));

	int recvNum = inputData.Nreceivers_;
	


    	int size3 = inputData.discretizationSize_[0] *
                inputData.discretizationSize_[1] *
                inputData.discretizationSize_[2];



    	InputDataOnDevice hostDataCopy = {(inputData.sourcePos_),
                                      (float) (2*3.141592f*inputData.f_),
                                      thrust::complex<float> (0, (float) (2*3.141592f*inputData.f_/inputData.c_)),
                                     (inputData.anomalyPos_),
                                      (inputData.anomalySize_),
                                      inputData.discretizationSize_,
                                      inputData.discreteBlockSize_,
                                      size3,
                                      inputData.discretizationSize_[0] *
                                      inputData.discretizationSize_[1],
                                      inputData.discretizationSize_[0],
                                      (float)(4*3.141592f*3.141592f*inputData.f_*inputData.f_*
                                      inputData.discreteBlockSize_[0]*inputData.discreteBlockSize_[1]*inputData.discreteBlockSize_[2])};


    hipMemcpy (deviceInputData, &hostDataCopy, sizeof (InputDataOnDevice), hipMemcpyHostToDevice);

    printf ("About to call kernel\n");
    DevicePrintData<<<1, 1>>> (deviceInputData);
    hipDeviceSynchronize ();
    printf ("Kernel returned\n");


    thrust::host_vector<thrust::complex<float> > hostDs2Matrix (size3);

    for (int x = 0; x < inputData.discretizationSize_[0]; x++)
    {
        for (int y = 0; y < inputData.discretizationSize_[1]; y++)
        {
            for (int z = 0; z < inputData.discretizationSize_[2]; z++)
            {
                int currentIndex = (x + y*inputData.discretizationSize_[0] + z*inputData.discretizationSize_[0]*inputData.discretizationSize_[1]);
                hostDs2Matrix[currentIndex] = thrust::complex<float> (float (inputData.ds2_[currentIndex]), 0.0);
            }
        }
    }

    thrust::device_vector<thrust::complex<float> > dS (hostDs2Matrix);


	thrust::device_vector <thrust::complex <float> > Ui (size3);
	 void * tempPtr = Ui.data ().get ();
    hipMemcpyToSymbol(HIP_SYMBOL(UiPtr),
                       &tempPtr,
                       sizeof(void*));


thrust::device_vector <Point3DDevice_t <float> > Points (size3);
	tempPtr = Points.data ().get ();
    hipMemcpyToSymbol(HIP_SYMBOL(PointsPtr),
                       &tempPtr,
                       sizeof(void*));
	
	

	//thrust::device_vector <Point3DDevice_t <float> > Points (size3); // remove ds or ui
	
	thrust::tabulate(Points.begin(), Points.end(), IndexFromSequence()); // filling Point with coordinates

	thrust::transform(dS.begin(), dS.end(), Points.begin(), Ui.begin(), UiMultiply()); // filling Ui array with w^2 * G(r) * ds^2 * h^3

	thrust::device_vector <thrust::complex <float> > d_output(recvNum);
	

	for (int i = 0; i < recvNum; i ++)
	{
		Point3D_t rj = inputData.receivers_[i];

		thrust::device_vector <int> BornForReciever(size3);
		thrust::complex <float> init = {};
		thrust::sequence(BornForReciever.begin(), BornForReciever.end()); 

		//float init = 0; //ui to global
		complexPlus binary_op;

		d_output [i] = thrust::transform_reduce(BornForReciever.begin(), BornForReciever.end(), BornCalculation (rj), init, binary_op); //born calc to global ui
	}

	thrust::host_vector<thrust::complex <float> > h_output(d_output);
	//*retData = h_output;

	for (int i = 0; i < recvNum; i++)
	{
		printf("%f + %fi\n", h_output[i].real(), h_output[i].imag());
	}
	return 0;
}

