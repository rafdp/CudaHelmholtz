#include "hip/hip_runtime.h"

#include "Builder.h"
#include <thrust/device_vector.h>
#include <thrust/transform.h>
#include <thrust/sequence.h>
#include <thrust/copy.h>
#include <thrust/fill.h>
#include <thrust/replace.h>
#include <thrust/complex.h>
#include <thrust/transform_reduce.h>
#include <thrust/transform.h>
#include <thrust/functional.h>

__constant__ thrust::complex<float> d_I_;
__constant__ float d_PI_;

template <typename T>
struct Point3DDevice_t
{
    T x, y, z;

    Point3DDevice_t (const Point3D_t &copy)
    {
        x = (T) copy.x;
        y = (T) copy.y;
        z = (T) copy.z;
    }

    template <typename T1, typename T2, typename T3>
    Point3DDevice_t (T1 tx, T2 ty, T3 tz) : 
        x (tx), y (ty), z (tz){}

    __host__ __device__
    T len () const
    {
        return (T) sqrtf (x*x + y*y + z*z);
    }
};

struct InputDataOnDevice
{
    Point3DDevice_t<float> sourcePos_;
    float w_; //DROP
    thrust::complex<float> uiCoeff_;
    Point3DDevice_t<float> anomalyPos_;
    Point3DDevice_t<float> anomalySize_;
    Point3DDevice_t<int>   discretizationSize_;
    Point3DDevice_t<int>   discreteBlockSize_;
    int                    size3_;
    int                    size2_;
    int                    size1_;
    float                  w2h3_;
};

__device__ InputDataOnDevice * inputDataPtr;

struct BornCalculation  // rewrite inputdata use
{
	const Point3DDevice_t <float> rj;
	BornCalculation(Point3D_t _rj) : rj(_rj) {}

    __host__ __device__
	thrust::complex<float> operator()(const thrust::complex <float> Ui, const Point3DDevice_t <float> r) const
	{

		InputDataOnDevice* d_inputData = inputDataPtr;

		Point3DDevice_t <float> dr = {r.x - rj.x, r.y - rj.y, r.z - rj.z};
						            
		return Ui * thrust::exp(d_inputData -> uiCoeff_ * dr.len()) / (4 * d_PI_ * dr.len());
	}
};

struct UiMultiply
{
	__host__ __device__
	thrust::complex <float> operator()(const thrust::complex<float>& ds, const Point3DDevice_t<float>& r) const 
	{
		InputDataOnDevice* d_inputData = inputDataPtr;
		return d_inputData -> w2h3_ * ds * thrust::exp(d_inputData -> uiCoeff_ * r.len()) / (4 * d_PI_ * r.len());
	}

};

struct IndexFromSequence
{
	__host__ __device__
		Point3DDevice_t <float> operator() (int idx) const
	{
		Point3DDevice_t<float> point = { idx % inputDataPtr->size2_,
			(idx / inputDataPtr->size1_) % inputDataPtr->discretizationSize_.y,
			idx / inputDataPtr->size2_ };
		point = { point.x*inputDataPtr->discreteBlockSize_.x*1.0f + inputDataPtr->anomalyPos_.x + inputDataPtr->discreteBlockSize_.x / 2.0,
			      point.y*inputDataPtr->discreteBlockSize_.y*1.0f + inputDataPtr->anomalyPos_.y + inputDataPtr->discreteBlockSize_.y / 2.0,
			      point.z*inputDataPtr->discreteBlockSize_.z*1.0f + inputDataPtr->anomalyPos_.z + inputDataPtr->discreteBlockSize_.z / 2.0 };
		return point;
	}
};

__global__

void DevicePrint ()
{
    printf ("--------------------------------------------------------------\n");
    printf ("threadIdx.x: %d\n", threadIdx.x);
    printf ("--------------------------------------------------------------\n");
}


__global__ void DevicePrintData (InputDataOnDevice * inputDataPtr)
{
    /*    Point3DDevice_t<float> sourcePos_;
    float w_; //DROP
    thrust::complex<float> uiCoeff_;
    Point3DDevice_t<float> anomalyPos_;
    Point3DDevice_t<float> anomalySize_;
    Point3DDevice_t<int>   discretizationSize_;
    Point3DDevice_t<int>   discreteBlockSize_;
    int                    size3_;
    int                    size2_;
    int                    size1_;
    float                  w2h3_;*/

    printf ("device address %x\n", inputDataPtr);

    InputDataOnDevice& handle = *(inputDataPtr);

    printf ("--------------------------------------------------------------\n");
    printf ("Printing from device:\n");
    printf ("   sourcePos_: %f %f %f\n",
            inputDataPtr->sourcePos_.x,
            handle.sourcePos_.y,
            handle.sourcePos_.z);
    printf ("   w: %f\n",
            handle.w_);
    printf ("   uiCoeff: %f i*%f\n",
            handle.uiCoeff_.real (),
            handle.uiCoeff_.imag ());
    printf ("   anomalySize_: %f %f %f\n",
            handle.anomalySize_.x,
            handle.anomalySize_.y,
            handle.anomalySize_.z);
    printf ("   discretizationSize_: %d %d %d\n",
            handle.discretizationSize_.x,
            handle.discretizationSize_.y,
            handle.discretizationSize_.z);
    printf ("   discreteBlockSize_: %d %d %d\n",
            handle.discreteBlockSize_.x,
            handle.discreteBlockSize_.y,
            handle.discreteBlockSize_.z);
    printf ("   size3_: %d %d %d\n", handle.size3_);
    printf ("   size2_: %d %d %d\n", handle.size2_);
    printf ("   size1_: %d %d %d\n", handle.size1_);
    printf ("   w2h3_: %f\n", handle.w2h3_);
    printf ("End print from device\n");
    printf ("--------------------------------------------------------------\n");
}


int main()
{
	InputData_t inputData = {};
	inputData.LoadData(); //pi and is



	InputDataOnDevice* deviceInputData = nullptr;

	printf ("ERROR: %s\n", hipGetErrorString(hipMalloc ((void**) &deviceInputData, sizeof (InputDataOnDevice))));

    printf ("ERROR: %s\n", hipGetErrorString(hipMemcpyToSymbol(HIP_SYMBOL(&inputData),
                                                                 &deviceInputData,
                                                                 sizeof(InputDataOnDevice*))));

	int recvNum = inputData.Nreceivers_;
	


    int size3 = inputData.discretizationSize_[0] *
                inputData.discretizationSize_[1] *
                inputData.discretizationSize_[2];

    #define PointConversion(var, type)\
    (Point3DDevice_t<type>) \
    {(type)(inputData.var.x),  \
     (type)(inputData.var.y), \
     (type)(inputData.var.z)}

    InputDataOnDevice hostDataCopy = {PointConversion (sourcePos_, float),
                                      (float) (2*3.141592f*inputData.f_),
                                      thrust::complex<float> (0, (float) (2*3.141592f*inputData.f_/inputData.c_)),
                                      PointConversion (anomalyPos_, float),
                                      PointConversion (anomalySize_, float),
                                      (Point3DDevice_t<int>){inputData.discretizationSize_[0],
                                       inputData.discretizationSize_[1],
                                       inputData.discretizationSize_[2]},
                                      (Point3DDevice_t<int>){inputData.discreteBlockSize_[0],
                                       inputData.discreteBlockSize_[1],
                                       inputData.discreteBlockSize_[2]},
                                      size3,
                                      inputData.discretizationSize_[0] *
                                      inputData.discretizationSize_[1],
                                      inputData.discretizationSize_[0],
                                      (float)(4*3.141592f*3.141592f*inputData.f_*inputData.f_*
                                      inputData.discreteBlockSize_[0]*inputData.discreteBlockSize_[1]*inputData.discreteBlockSize_[2])};

    #undef PointConversion

    hipMemcpy (deviceInputData, &hostDataCopy, sizeof (InputDataOnDevice), hipMemcpyHostToDevice);

    printf ("About to call kernel\n");
    DevicePrintData<<<1, 1>>> (deviceInputData);
    hipDeviceSynchronize ();
    printf ("Kernel returned\n");


    thrust::host_vector<thrust::complex<float> > hostDs2Matrix (size3);

    for (int x = 0; x < inputData.discretizationSize_[0]; x++)
    {
        for (int y = 0; y < inputData.discretizationSize_[1]; y++)
        {
            for (int z = 0; z < inputData.discretizationSize_[2]; z++)
            {
                int currentIndex = (x + y*inputData.discretizationSize_[0] + z*inputData.discretizationSize_[0]*inputData.discretizationSize_[1]);
                hostDs2Matrix[currentIndex] = thrust::complex<float> (float (inputData.ds2_[currentIndex]), 0.0);
            }
        }
    }

    thrust::device_vector<thrust::complex<float> > dS (hostDs2Matrix);


	thrust::device_vector <thrust::complex <float> > Ui (size3);
	thrust::device_vector <Point3DDevice_t <float> > Points (size3); // remove ds or ui
	
	thrust::tabulate(Points.begin(), Points.end(), IndexFromSequence()); // filling Point with coordinates

	thrust::transform(dS.begin(), dS.end(), Points.begin(), Ui.begin(), UiMultiply()); // filling Ui array with w^2 * G(r) * ds^2 * h^3

	thrust::device_vector <thrust::complex <float> > d_output(recvNum);
	thrust::host_vector   <thrust::complex <float> > h_output(recvNum);

	for (int i = 0; i < recvNum; i ++)
	{
		Point3D_t rj = inputData.receivers_[i];

		thrust::device_vector <thrust::complex<float>> BornForReciever(size3);
		//thrust::sequence(BornForReciever.begin(), BornForReciever.end()); 

		float init = 0; //ui to global
		thrust::plus <float> binary_op;

		d_output [i] = thrust::transform_reduce(Points.begin(), Points.end(), BornCalculation (rj), init, binary_op); //born calc to global ui
	}

	h_output = d_output;

	for (int i = 0; i < recvNum; i++)
	{
		printf("%f + %fi\n", h_output[i].real(), h_output[i].imag());
	}

	return 0;
}

